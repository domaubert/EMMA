

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern "C" void initlocaldevice(int,int);
extern "C" int countdevices(int);
extern "C" void checkdevice(int);
extern "C" void CPU2GPU(float *gpupt, float *cpupt, int noctet);
extern "C" void GPU2CPU(float *cpupt, float *gpupt, int noctet);
extern "C" void GPU2GPU(float *cpupt, float *gpupt, int noctet);
extern "C" void CPU2GPU_INT(int *gpupt, int *cpupt, int noctet);
extern "C" void GPU2CPU_INT(int *cpupt, int *gpupt, int noctet);
extern "C" void CPU2GPU_UINT(unsigned int *gpupt, unsigned int *cpupt, int noctet);
extern "C" void GPU2CPU_UINT(unsigned int *gpupt, unsigned int *cpupt, int noctet);

//#define CUERR() printf("\n %s \n",cudaGetErrorString(cudaGetLastError()))

//************************************************************************************************************************
//************************************************************************************************************************

void CPU2GPU(float *gpupt, float *cpupt, int noctet)
{
  hipMemcpy(gpupt,cpupt,noctet,hipMemcpyHostToDevice);  
}

void GPU2CPU(float *cpupt, float *gpupt, int noctet)
{
  hipMemcpy(cpupt,gpupt,noctet,hipMemcpyDeviceToHost);  
}

void GPU2GPU(float *cpupt, float *gpupt, int noctet)
{
  hipMemcpy(cpupt,gpupt,noctet,hipMemcpyDeviceToDevice);  
}

void CPU2GPU_INT(int *gpupt, int *cpupt, int noctet)
{
  hipMemcpy(gpupt,cpupt,noctet,hipMemcpyHostToDevice);  
}

void GPU2CPU_INT(int *cpupt, int *gpupt, int noctet)
{
  hipMemcpy(cpupt,gpupt,noctet,hipMemcpyDeviceToHost);  
}

void CPU2GPU_UINT(unsigned int *gpupt, unsigned int *cpupt, int noctet)
{
  hipMemcpy(gpupt,cpupt,noctet,hipMemcpyHostToDevice);  
}

void GPU2CPU_UINT(unsigned int *cpupt, unsigned int *gpupt, int noctet)
{
  hipMemcpy(cpupt,gpupt,noctet,hipMemcpyDeviceToHost);  
}

int countdevices(int rank)
{
  int count;
  hipGetDeviceCount(&count);
  //printf("%d device(s) found locally by proc %d\n",count,rank);
  return count;
}

void initlocaldevice(int rank, int count)
{
  int devicenum=rank%count;
  struct hipDeviceProp_t prop;
  hipSetDevice(devicenum);
  hipGetDeviceProperties(&prop,devicenum);
  printf("Local Device #%d initialized for proc #%d with %d B of VRAM\n",devicenum,rank,(int)prop.totalGlobalMem);
}


void checkdevice(int rank)
{
  int idevice;
  hipGetDevice(&idevice);
  printf("Local Device #%d dedicated to proc #%d\n",idevice,rank);
}

