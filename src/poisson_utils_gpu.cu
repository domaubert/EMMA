#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "prototypes.h"
#include <mpi.h>
//#include <cudpp.h>
#include "gpu_type.h"

#ifdef WGRAV

extern "C" struct OCT *gatherstencilgrav(struct OCT *octstart, struct GGRID *stencil, int stride, struct CPUINFO *cpu, int *nread);
extern "C" struct OCT *scatterstencilgrav(struct OCT *octstart, struct STENGRAV *stencil, int nread, int stride, struct CPUINFO *cpu);
extern "C" void clean_vecpos(int level,struct OCT **firstoct);
extern "C" struct OCT *gatherstencilgrav_nei(struct OCT *octstart, struct STENGRAV *gstencil, int stride, struct CPUINFO *cpu, int *nread);
extern "C" void update_pot_in_tree(int level,struct OCT ** firstoct,  struct CPUINFO *cpu, struct RUNPARAMS *param, REAL *distout, REAL *normpout);
extern "C" REAL PoissonJacobiGPU(int level,struct RUNPARAMS *param, struct OCT ** firstoct,  struct CPUINFO *cpu, struct STENGRAV *stencil, int stride, REAL tsim);
extern "C" void create_gravstencil_GPU(struct CPUINFO *cpu, int stride);
extern "C" void create_pinned_gravstencil(struct STENGRAV *gstencil, int stride);
extern "C" void destroy_pinned_gravstencil(struct STENGRAV *gstencil, int stride);
extern "C" void destroy_gravstencil_GPU(struct CPUINFO *cpu, int stride);
extern "C" void mpi_exchange_level(struct CPUINFO *cpu, struct PACKET **sendbuffer, struct PACKET **recvbuffer, int field, int cmp_keys, int level);



// =======================================================
// =======================================================

void create_pinned_gravstencil(struct STENGRAV *gstencil, int stride){
  struct GGRID *grav_stencil;
  hipHostMalloc( (void**)&grav_stencil, sizeof(struct GGRID)*stride );
  gstencil->stencil=grav_stencil;
  REAL *pnew;
  REAL *res;
  REAL *resLR;
  hipHostMalloc( (void**)&pnew, sizeof(REAL)*stride*8 );
  hipHostMalloc( (void**)&res, sizeof(REAL)*stride*8 );
  gstencil->pnew=pnew;
  gstencil->res=res;
#ifdef ONFLYRED
  hipHostMalloc( (void**)&resLR, sizeof(REAL)*stride);
  gstencil->resLR=resLR;
#endif
  //abort();
}

// ===============================================================
// ===============================================================

void destroy_pinned_gravstencil(struct STENGRAV *gstencil, int stride){

  hipHostFree(gstencil->stencil);
  hipHostFree(gstencil->pnew);
  hipHostFree(gstencil->res);
#ifdef ONFLYRED
  hipHostFree(gstencil->resLR);
#endif
}


// ===============================================================
// ===============================================================


void create_gravstencil_GPU(struct CPUINFO *cpu, int stride){
  hipMalloc((void **)&(cpu->dev_stencil),sizeof(struct GGRID)*stride);
  hipMalloc((void **)&(cpu->res),sizeof(REAL)*stride*8);
  hipMalloc((void **)&(cpu->pnew),sizeof(REAL)*stride*8);
  hipMalloc((void **)&(cpu->resLR),sizeof(REAL)*stride);
}

void destroy_gravstencil_GPU(struct CPUINFO *cpu, int stride){
  hipFree(cpu->dev_stencil);
  hipFree(cpu->res);
  hipFree(cpu->pnew);
  hipFree(cpu->resLR);

}

// =======================================================
__device__ void getcellnei_gpu(int cindex, int *neip, int *cell)
{
  switch(cindex){
  case 0:
    neip[0]=0;cell[0]=1;
    neip[1]=6;cell[1]=1;
    neip[2]=2;cell[2]=2;
    neip[3]=6;cell[3]=2;
    neip[4]=4;cell[4]=4;
    neip[5]=6;cell[5]=4;
    break;
  case 1:
    neip[0]=6;cell[0]=0;
    neip[1]=1;cell[1]=0;
    neip[2]=2;cell[2]=3;
    neip[3]=6;cell[3]=3;
    neip[4]=4;cell[4]=5;
    neip[5]=6;cell[5]=5;
    break;
  case 2:
    neip[0]=0;cell[0]=3;
    neip[1]=6;cell[1]=3;
    neip[2]=6;cell[2]=0;
    neip[3]=3;cell[3]=0;
    neip[4]=4;cell[4]=6;
    neip[5]=6;cell[5]=6;
    break;
  case 3:
    neip[0]=6;cell[0]=2;
    neip[1]=1;cell[1]=2;
    neip[2]=6;cell[2]=1;
    neip[3]=3;cell[3]=1;
    neip[4]=4;cell[4]=7;
    neip[5]=6;cell[5]=7;
    break;
  case 4:
    neip[0]=0;cell[0]=5;
    neip[1]=6;cell[1]=5;
    neip[2]=2;cell[2]=6;
    neip[3]=6;cell[3]=6;
    neip[4]=6;cell[4]=0;
    neip[5]=5;cell[5]=0;
    break;
  case 5:
    neip[0]=6;cell[0]=4;
    neip[1]=1;cell[1]=4;
    neip[2]=2;cell[2]=7;
    neip[3]=6;cell[3]=7;
    neip[4]=6;cell[4]=1;
    neip[5]=5;cell[5]=1;
    break;
  case 6:
    neip[0]=0;cell[0]=7;
    neip[1]=6;cell[1]=7;
    neip[2]=6;cell[2]=4;
    neip[3]=3;cell[3]=4;
    neip[4]=6;cell[4]=2;
    neip[5]=5;cell[5]=2;
    break;
  case 7:
    neip[0]=6;cell[0]=6;
    neip[1]=1;cell[1]=6;
    neip[2]=6;cell[2]=5;
    neip[3]=3;cell[3]=5;
    neip[4]=6;cell[4]=3;
    neip[5]=5;cell[5]=3;
    break;
  }

}


//========================================================================
//========================================================================

__global__ void dev_PoissonJacobi_single(struct GGRID *stencil, int level, int curcpu, int nread,int stride,REAL dx, int flag, REAL factdens, REAL *vres, REAL *stockres, REAL *stockpnew, REAL *stockresLR){

  // flag=1 means the residual contains the norm of the density
  // flag=0 means the resiual contains the actual residual of the Poisson Equation

  int inei,icell;
  int i;
  REAL temp;
  REAL res;
  int vnei[6],vcell[6];
  //int ioct[7]={12,14,10,16,4,22,13};
  int ioct[7]={0,1,2,3,4,5,6};
  struct Gtype *curcell;
  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;

  i=bx*blockDim.x+tx;


  stockresLR[i]=0.;

  for(icell=0;icell<8;icell++){ // we scan the cells
    getcellnei_gpu(icell, vnei, vcell); // we get the neighbors

    temp=0.;
    res=0.;

    curcell=&(stencil[i].oct[ioct[6]].cell[icell].gdata);

    // Computing the laplacian ===========================

    for(inei=0;inei<6;inei++){
      temp+=stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].gdata.p;
    }

    // setting up the residual
    res=temp;

    // we finish the laplacian
    stockpnew[icell+i*8]=(temp-dx*dx*curcell->d*factdens)/6.0;

    // we finsih the residual
    res=(res-6.0*curcell->p)/(dx*dx)-factdens*curcell->d;

    // low res
    stockresLR[i]+=res*0.125;

    // we store the local residual
    if(flag) {
      vres[icell+i*8]=factdens*curcell->d*factdens*curcell->d;
      stockres[icell+i*8]=factdens*curcell->d;
    }
    else{
      vres[icell+i*8]=res*res;
      stockres[icell+i*8]=res;
    }

  }


}

//=========================================================================================================

REAL PoissonJacobiGPU(int level,struct RUNPARAMS *param, struct OCT ** firstoct, struct CPUINFO *cpu, struct STENGRAV *stencil, int stride, REAL tsim)
{
  REAL dxcur;
  int iter;
  struct OCT *nextoct;
  struct OCT *curoct;
  struct OCT *curoct0;

  int nreadtot;
  int nread;
  REAL residual,dres;
  //  int icell;
  int nitmax;
  REAL factdens;
  REAL fnorm,res0=0.;
  REAL *rloc;
  REAL *resA;
  REAL *resB;
  REAL dist,normp,dresconv;
  int crit;
  int ng;
  int nt;

  CUDA_CHECK_ERROR("Poisson Start");
  // ======================= some stuf for CUDPP =======================/

  resA= cpu->gresA;
  resB= cpu->gresB;

  hipHostMalloc((void**)&rloc,sizeof(REAL)*cpu->nstream);
  /* hipMalloc((void **)&resA,sizeof(REAL)*stride*8); */
  /* hipMalloc((void **)&resB,sizeof(REAL)*stride*8); */

  /* printf("POINT IN GPU=%p\n",resA); */
  /* printf("POINT IN GPU=%p\n",resB); */


  /* struct CUPARAM cuparam; */

  /* cudppCreate(&(cuparam.theCudpp)); */

  /* cuparam.config.algorithm = CUDPP_SCAN; */
  /* cuparam.config.datatype = CUDPP_DOUBLE; */
  /* cuparam.config.options=CUDPP_OPTION_BACKWARD | CUDPP_OPTION_INCLUSIVE; */
  /* cuparam.config.op=CUDPP_ADD; */

  /* cuparam.scanplan =0; */
  //cudppPlan(cuparam.theCudpp,&(cuparam.scanplan), cuparam.config, stride*8, 1, 0);
  //printf("3 Start Error Grav =%s nreadtot=%d\n",hipGetErrorString(hipGetLastError()),nreadtot);

  // THE PLAN
  struct CUPARAM *cuparam;
  cuparam=(struct CUPARAM *)cpu->cuparam;


  //======================== END CUDPP STUFF ========================/

  int is,offset;

  int nstream=cpu->nstream;
  /* hipStream_t stream[cpu->nstream]; */
  /* int vnread[cpu->nstream]; */
  hipStream_t *stream;
  int *vnread;

  /* if(cpu->nthread*cpu->nstream>stride){ */

  /* } */

  hipHostMalloc((void**)&stream,sizeof(hipStream_t)*nstream);
  hipHostMalloc((void**)&vnread,sizeof(int)*nstream);

  // creating the streams
  for(is=0;is<nstream;is++){
    hipStreamCreate(&stream[is]);
  }



  // Computing the factor of the density
  if(level>=param->lcoarse){
#ifndef TESTCOSMO
    factdens=4.0*M_PI;
#else
    //    factdens=6.0*tsim; WARNING JUST TESTING WITHOUT TSIM!!!
    factdens=6.0;
#endif
  }
  else{
    factdens=1.;
  }

  // Computing the max number for iteration

  if((level==param->mgridlmin)||(level>param->lcoarse)){
    nitmax=param->niter;
  }
  else{
    nitmax=param->nrelax;
  }

  dxcur=POW(0.5,level);


  // Scanning the Octs

  fnorm=0.;
  for(iter=0;iter<nitmax;iter++){
    // --------------- some inits for iterative solver
    residual=0.;

    // --------------- setting the first oct of the level
    nextoct=firstoct[level-1];
    nreadtot=0;
    if((nextoct!=NULL)&&(cpu->noct[level-1]!=0)){
      do {
	curoct0=nextoct;
	curoct=curoct0;


	// streaming data
	offset=0;
	for(is=0;is<nstream;is++){

	  // ------------ gathering the stencil value values

	  curoct=nextoct;
	  if(curoct!=NULL){
	    nextoct=gatherstencilgrav(curoct,stencil->stencil+offset,stride/nstream,cpu, vnread+is);
	    if(vnread[is]!=0){
	      ng=((vnread[is]-1)/cpu->nthread)+1; // +1 is for leftovers

	    if(ng==1){
	      nt=vnread[is];
	    }
	    else{
	      nt=cpu->nthread;
	    }

	    dim3 gridoct(ng);
	    dim3 blockoct(nt);
	    hipMemcpyAsync(cpu->dev_stencil+offset,stencil->stencil+offset,vnread[is]*sizeof(struct GGRID),hipMemcpyHostToDevice,stream[is]);
#ifndef NOCOMP
	    // ------------ solving the hydro
	    dev_PoissonJacobi_single<<<gridoct,blockoct,0,stream[is]>>>(cpu->dev_stencil+offset,level,cpu->rank,vnread[is],stride,dxcur,(iter==0),factdens,resA+offset*8,cpu->res+offset*8,cpu->pnew+offset*8,cpu->resLR+offset);
	    // ------------ computing the residuals

	    hipStreamSynchronize(stream[is]);
	    //cudppScan(cuparam->scanplan, resB+offset*8, resA+offset*8, vnread[is]*8);
#endif

	    hipMemcpyAsync(rloc+is,resB+offset*8,sizeof(REAL),hipMemcpyDeviceToHost,stream[is]);

	    hipMemcpyAsync(stencil->res+offset*8,cpu->res+offset*8,(vnread[is])*sizeof(REAL)*8,hipMemcpyDeviceToHost,stream[is]);
	    hipMemcpyAsync(stencil->pnew+offset*8,cpu->pnew+offset*8,(vnread[is])*sizeof(REAL)*8,hipMemcpyDeviceToHost,stream[is]);
	    hipMemcpyAsync(stencil->resLR+offset,cpu->resLR+offset,(vnread[is])*sizeof(REAL),hipMemcpyDeviceToHost,stream[is]);

	    offset+=vnread[is];
	    //residual=(residual>rloc[is]?residual:rloc[is])*(iter!=0);
 	    }
	  }
	}



	// ------------ scatter back the data
	hipDeviceSynchronize();
	for(is=0;is<cpu->nstream;is++) {
	  if(iter==0){
	    //printf("rloc=%e\n",rloc[is]);
	    fnorm+=rloc[is]*(vnread[is]>0);
	  }
	  else{
	    residual+=rloc[is]*(vnread[is]>0);
	  }
	}

	nread=offset;
	nextoct=scatterstencilgrav(curoct0,stencil, nread, stride,cpu);

	nreadtot+=nread;

      }while(nextoct!=NULL);
    }


    // at this stage an iteration has been completed : let's update the potential and compute the residual

    if(nreadtot>0){
      update_pot_in_tree(level,firstoct,cpu,param,&dist,&normp);
    }

#ifdef WMPI
    //printf("iter=%d\n",iter);
    if((iter<=param->niter)||(iter%1==0)){
      mpi_exchange_level(cpu,cpu->sendbuffer,cpu->recvbuffer,2,(iter==0),level); // potential field exchange
      if(iter==0){
	//if(level==7) printf("rank=%d fnorm=%e\n",cpu->rank,fnorm);
	MPI_Allreduce(MPI_IN_PLACE,&fnorm,1,MPI_REEL,MPI_SUM,MPI_COMM_WORLD);
      }
      else{
	MPI_Allreduce(MPI_IN_PLACE,&residual,1,MPI_REEL,MPI_SUM,MPI_COMM_WORLD);
	MPI_Allreduce(MPI_IN_PLACE,&dist,1,MPI_REEL,MPI_SUM,MPI_COMM_WORLD);
	MPI_Allreduce(MPI_IN_PLACE,&normp,1,MPI_REEL,MPI_SUM,MPI_COMM_WORLD);
      }
    }
#endif


    if((iter==1)&&(level>=param->lcoarse)) res0=residual;

    if(iter>0){

      // here we test the convergence of the temporary solution
      dresconv=SQRT(dist/normp);

      // here we test the zero level of Poisson equation
      if(level<param->lcoarse){
	dres=SQRT(residual);
      }
      else{
	dres=SQRT(residual/fnorm);
      }

      // we take the smallest
      dres=(dres<dresconv?dres:dresconv);
      crit=(dres<dresconv?0:1);

      if((dres)<param->poissonacc){
	if(level>=param->lcoarse) break;
      }
    }

  }


  if(level>param->lcoarse){
    if(cpu->rank==RANK_DISP) printf("GPU | level=%d iter=%d res=%e fnorm=%e\n",level,iter,dres,fnorm);
  }
  else{
    if(cpu->rank==RANK_DISP) printf("GPU | level=%d iter=%d res=%e fnorm=%e resraw=%e res0=%e crit=%d\n",level,iter,dres,fnorm,SQRT(residual),SQRT(res0),crit);
  }


  //  printf("GPU | level=%d iter=%d res=%e tgat=%e tcal=%e tscat=%e tall=%e tup=%e tglob=%e\n",level,iter,dres,tgat/iter,tcal/iter,tscat/iter,tall/iter,tup/iter,tglob/iter);
  //printf("GPU | level=%d iter=%d res=%e \n",level,iter,dres);

  /* cudppDestroyPlan(cuparam.scanplan); */
  /* cudppDestroy(cuparam.theCudpp); */
  /* hipFree(resA); */
  /* hipFree(resB); */
  hipHostFree(rloc);

  // Destroying the streams
  for(is=0;is<nstream;is++){
    hipStreamDestroy(stream[is]);
  }



  hipHostFree(stream);
  hipHostFree(vnread);

  //printf("Start Error Grav =%s nreadtot=%d\n",hipGetErrorString(hipGetLastError()),nreadtot);
  CUDA_CHECK_ERROR("Poisson Stop");
  return dres;
}


#endif
