
#include <hip/hip_runtime.h>

#ifdef WRAD
#ifdef WCHEM

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "prototypes.h"
#include "oct.h"
#include <string.h>
#include <mpi.h>
//#include "atomic_data/Atomic.h"
#include "gpu_type.h"


#define FRAC_VAR (0.1)
#define idloc 0

//================================================================================
__device__ void dE2T(struct Rtype *R, REAL aexp,struct RUNPARAMS *param){

  REAL tloc;
  REAL eint=R->eint;
  REAL nH=R->nh;
  REAL x=R->nhplus/R->nh;
  REAL pstar=param->unit.unit_n*param->unit.unit_d*POW(param->unit.unit_v,2);
  nH=nH/POW(aexp,3)*param->unit.unit_N;
  eint=eint/POW(aexp,5)*pstar;
  tloc=eint/(1.5*nH*KBOLTZ*(1.+x));
  R->temp=tloc;
}


// ============================================================================
__device__ REAL dcucompute_alpha_b(REAL temp, REAL unit_number, REAL aexp)
{
  // CASE B recombination rate m**3 s*-1
  // temperature should be given in Kelvin
  
  REAL alpha_b,lambda;
  lambda=2e0*157807e0/temp;
  alpha_b=2.753e-14*POW(lambda,1.5)/POW(1e0+POW(lambda/2.740,0.407),2.242); //cm3/s
#ifdef TESTCOSMO
  alpha_b=alpha_b*1e-6*unit_number;///(aexp*aexp*aexp); //m3/s
#else
  alpha_b=alpha_b*1e-6*unit_number; //m3/s
#endif
  return alpha_b;
}

//=========================================================
//=========================================================

__device__ REAL dcucompute_alpha_a(REAL temp, REAL unit_number, REAL aexp)
{
  // CASE A recombination rate m**3 s*-1
  // temperature should be given in Kelvin
  
  REAL alpha_a,lambda;
  lambda=2e0*157807e0/temp;
  alpha_a=1.269e-13*POW(lambda,1.503)/POW(1e0+POW(lambda/0.522,0.470),1.923); //cm3/s
#ifdef TESTCOSMO
  alpha_a=alpha_a*1e-6*unit_number;///(aexp*aexp*aexp); //m3/s
#else
  alpha_a=alpha_a*1e-6*unit_number; //m3/s
#endif
  return alpha_a;
}

//=========================================================
//=========================================================

__device__ REAL dcucompute_beta(REAL temp, REAL unit_number, REAL aexp)
{
  // Collizional ionization rate m**3 s*-1
  // temperature in Kelvin
  REAL beta,T5;
  T5=temp/1e5;
  beta=5.85e-11*SQRT(temp)/(1+SQRT(T5))*EXP(-(157809e0/temp)); //cm3/s
#ifdef TESTCOSMO
  beta=beta*1e-6*unit_number;///(aexp*aexp*aexp); // !m3/s
#else
  beta=beta*1e-6*unit_number; // !m3/s
#endif
  return beta;
}

//**********************************************************************************
//**********************************************************************************
__device__ void dcuCompCooling(REAL temp, REAL x, REAL nH, REAL *lambda, REAL *tcool, REAL aexp,REAL CLUMPF)
{

  REAL c1,c2,c3,c4,c5,c6;
  REAL unsurtc;
  REAL nh2;

  nh2=nH*1e-6;// ! m-3 ==> cm-3
  

  // Collisional Ionization Cooling

  c1=EXP(-157809.1e0/temp)*1.27e-21*SQRT(temp)/(1.+SQRT(temp/1e5))*x*(1.-x)*nh2*nh2*CLUMPF;
  

  // Case A Recombination Cooling

  c2=1.778e-29*temp*POW(2e0*157807e0/temp,1.965e0)/POW(1.+POW(2e0*157807e0/temp/0.541e0,0.502e0),2.697e0)*x*x*nh2*nh2*CLUMPF;
  
  
  // Case B Recombination Cooling

  c6=3.435e-30*temp*POW(2e0*157807e0/temp,1.970e0)/POW(1.+(POW(2e0*157807e0/temp/2.250e0,0.376e0)),3.720e0)*x*x*nh2*nh2*CLUMPF;
  c6=0.;

  // Collisional excitation cooling

  c3=EXP(-118348e0/temp)*7.5e-19/(1+SQRT(temp/1e5))*x*(1.-x)*nh2*nh2*CLUMPF;
  
  
  // Bremmsstrahlung

  c4=1.42e-27*1.5e0*SQRT(temp)*x*x*nh2*nh2*CLUMPF;
  
  // Compton Cooling
  
  /* c5=1.017e-37*POW(2.727/aexp,4)*(temp-2.727/aexp)*nh2*x; */
   c5=0.;
#ifndef WRADTEST
   c5=5.406e-24*(temp-2.727/aexp)/POW(aexp/0.001,4)*x*nh2;
  REAL Ta=2.727/aexp; c5=5.406e-36*(temp-Ta)/(aexp*aexp*aexp*aexp)*x*nh2;
#endif
  // Overall Cooling
  
  *lambda=c1+c2+c3+c4+c5+c6;// ! erg*cm-3*s-1
  

  // Unit Conversion

  *lambda=(*lambda)*1e-7*1e6;// ! J*m-3*s-1

  // cooling times

  unsurtc=FMAX(c1,c2);
  unsurtc=FMAX(unsurtc,c3);
  unsurtc=FMAX(unsurtc,c4);
  unsurtc=FMAX(unsurtc,FABS(c5));
  unsurtc=FMAX(unsurtc,c6)*1e-7;// ==> J/cm3/s

  *tcool=1.5e0*nh2*(1.+x)*KBOLTZ*temp/unsurtc; //Myr
}


// ===========================================================================================================================

__global__ void dchemrad(struct RGRID *stencil, int nread, int stride, struct CPUINFO *cpu, REAL dxcur, REAL dtnew, struct RUNPARAMS *param, REAL aexporg, int chemonly)
{
  int i,icell,igrp;
  //int idloc=0;
  int nitcool=0;

  REAL hnu0=13.6*1.6022e-19,
    Cool,
    tcool,
    dtcool,
    tcool1,
    currentcool_t=0.,
    alpha,
    alphab,
    beta,
    tloc,
    xt,
    eintt,
    ai_tmp1=0.,
    et[NGRP],
    p[NGRP];

  REAL aexp;
  REAL ebkg[NGRP];
  REAL z=1./aexporg-1.;

  REAL c=param->clightorg*LIGHT_SPEED_IN_M_PER_S; 			// switch back to physical velocity m/s

  REAL hnu[NGRP];
  REAL alphae[NGRP];
  REAL alphai[NGRP];
  REAL factgrp[NGRP];

  for(igrp=0;igrp<NGRP;igrp++) {
    hnu[igrp]=param->atomic.hnu[igrp];
    alphae[igrp]=param->atomic.alphae[igrp];
    alphai[igrp]=param->atomic.alphai[igrp];
    factgrp[igrp]=param->atomic.factgrp[igrp];
  }


#ifdef S_X
  REAL E0overI[NGRP];
  REAL N2[NGRP];
  REAL F2[NGRP];
#endif

#define BLOCKCOOL 1 // KEPT FROM CUDATON FOR SIMPLICITY
#define idloc3 0 // KEPT FROM CUDATON FOR SIMPLICITY

  REAL
    egyloc[BLOCKCOOL*NGRP],
    floc[3*BLOCKCOOL*NGRP],
    srcloc[BLOCKCOOL*NGRP],
    x0[BLOCKCOOL],
    nH[BLOCKCOOL],
    eint[BLOCKCOOL];


  REAL dt=dtnew*param->unit.unit_t*POW(aexporg,2);

  REAL emin;
  struct Rtype R;
  REAL fudgecool=param->fudgecool;
  int ncvgcool=param->ncvgcool;
  REAL E0;
#ifdef SCHAYE
  REAL navg=(param->cosmo->ob/param->cosmo->om)/(PROTON_MASS*MOLECULAR_MU)*param->unit.unit_d;
#endif
  REAL xorg;

  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
  i=bx*blockDim.x+tx;


  if(i<nread){
    for(icell=0;icell<8;icell++){ // we scan the cells

      if(stencil[i].oct[6].cell[icell].split) continue; // we dont treat split cells

      memcpy(&R,&stencil[i].New.cell[icell].rfieldnew,sizeof(struct Rtype));// We get the local physical quantities after transport update

#ifdef HOMOSOURCE
      // we override the value with the homogeneous source density
      R.src=param->bkg;
#endif


	  //if(eint[idloc]!=E0) printf("1!\n");
	  /// ==================== UV Background
#ifdef UVBKG
	  if(NGRP>1) printf("WARNING BAD BEHAVIOR FOR BKG with NGRP>1 !\n");
	  //for(igrp=0;igrp<NGRP;igrp++) ebkg[igrp]=3.6*(z<3?1.:4./(1+z))  ;  // Katz simple model

	  // Poor FIT to Haardt & MAdau 2012
  /*
	  for(igrp=0;igrp<NGRP;igrp++){
	    REAL amp=1.2e-16,sig=1.,zavg=2,mz=1e-18,pz=1.2e-17;
	    ebkg[igrp]=amp/(sig*SQRT(2*M_PI))*exp(-POW((z-zavg),2)/(2.*POW(sig,2)))+mz*z+pz; // comoving photons/s/m3
	  }
  */

#else
	  for(igrp=0;igrp<NGRP;igrp++) ebkg[igrp]=0.;
#endif

      // switch to physical units, chemistry remains unchanged with and without cosmo
      for (igrp=0;igrp<NGRP;igrp++)
	{
	  egyloc[idloc+igrp*BLOCKCOOL]   =R.e[igrp]/(aexporg*aexporg*aexporg)*param->unit.unit_N;//+ebkg[igrp];
	  floc[0+idloc3+igrp*BLOCKCOOL*3]=R.fx[igrp]/POW(aexporg,4)*param->unit.unit_l/param->unit.unit_t*param->unit.unit_N;
	  floc[1+idloc3+igrp*BLOCKCOOL*3]=R.fy[igrp]/POW(aexporg,4)*param->unit.unit_l/param->unit.unit_t*param->unit.unit_N;
	  floc[2+idloc3+igrp*BLOCKCOOL*3]=R.fz[igrp]/POW(aexporg,4)*param->unit.unit_l/param->unit.unit_t*param->unit.unit_N;
	}


      x0[idloc]=R.nhplus/R.nh;
      xorg= x0[idloc];
      nH[idloc]=R.nh/(aexporg*aexporg*aexporg)*param->unit.unit_N;


      eint[idloc]=R.eint/POW(aexporg,5)*param->unit.unit_n*param->unit.unit_d*POW(param->unit.unit_v,2);
      emin=PMIN/(GAMMA-1.)/POW(aexporg,5)*param->unit.unit_n*param->unit.unit_d*POW(param->unit.unit_v,2); // physical minimal pressure

      for (igrp=0;igrp<NGRP;igrp++){
      srcloc[idloc+igrp*BLOCKCOOL]=(R.src[igrp]*param->unit.unit_N/param->unit.unit_t/(aexporg*aexporg))/POW(aexporg,3); //phot/s/dv (physique)
      }

// R.src phot/unit_t/unit_dv (comobile)
      REAL eorg=eint[idloc];
      REAL etorg=egyloc[idloc];
      REAL torg=eint[idloc]/(1.5*nH[idloc]*KBOLTZ*(1.+x0[idloc]));

      //if(srcloc[0]>0) 	printf("nh=%e %e %e %e\n",R.nh,R.e[0],eint[idloc],3[idloc]);

      // at this stage we are ready to do the calculations

      // DEALING WITH CLUMPING ----------------------
#ifdef WCLUMP
      REAL CLUMPF2=FMIN(FMAX(POW(nH[idloc]/6.,0.7),1.),40.);
      REAL CLUMPI=1.;
#else
      REAL CLUMPF2=1.;
      REAL CLUMPI=1.;
#endif


      for(igrp=0;igrp<NGRP;igrp++)
	{
	  alphai[igrp] *= CLUMPI;
	  alphae[igrp] *= CLUMPI;
	}

      // -------------------------------------------------

      /// local cooling loop -------------------------------
      aexp=aexporg;
      fudgecool=param->fudgecool;
      currentcool_t=0.;
      nitcool=0.;
      REAL da;
      //printf("cpu=%d fudge=%e ncv=%d currentcool_t=%e dt=%e\n",cpu->rank,param->fudgecool,ncvgcool,currentcool_t,dt);

      // local cooling loop -------------------------------
      while(currentcool_t<dt)
	{


	  /// Cosmological Adiabatic expansion effects ==============
#ifdef TESTCOSMO
	  REAL hubblet=param->cosmo->H0*SQRT(param->cosmo->om/aexp+param->cosmo->ov*(aexp*aexp))/aexp*(1e3/(1e6*PARSEC)); // s-1 // SOMETHING TO CHECK HERE
#else
	  REAL hubblet=0.;
#endif


	  //if(eint[idloc]!=E0) printf("2!\n");
	  tloc=eint[idloc]/(1.5*nH[idloc]*KBOLTZ*(1.+x0[idloc]));

	  //== Getting a timestep
	  dcuCompCooling(tloc,x0[idloc],nH[idloc],&Cool,&tcool1,aexp,CLUMPF2);
	  ai_tmp1=0.;

	  //if(eint[idloc]!=E0) printf("3!\n");

	  if(fudgecool<1e-20){
	    printf("eint=%e(%e<%e) nH=%e x0=%e(%e) T=%e(%e) N=%e(%e)\n",eint[idloc],eorg,emin,nH[idloc],x0[idloc],xorg,tloc,torg,et[0],etorg);
	    //if(fudgecool<1e-20) abort();
	  }

	  for (igrp=0;igrp<NGRP;igrp++) ai_tmp1 += ((alphae[igrp])*hnu[igrp]-(alphai[igrp])*hnu0)*egyloc[idloc+igrp*BLOCKCOOL];

	  tcool=FABS(eint[idloc]/(nH[idloc]*(1.0-x0[idloc])*ai_tmp1*(!chemonly)-Cool));
	  ai_tmp1=0.;
	  dtcool=FMIN(fudgecool*tcool,dt-currentcool_t);

	  alpha=dcucompute_alpha_a(tloc,1.,1.)*CLUMPF2;
	  alphab=dcucompute_alpha_b(tloc,1.,1.)*CLUMPF2;
	  beta=dcucompute_beta(tloc,1.,1.)*CLUMPF2;

	  //== Update

	  // ABSORPTION
	  int test = 0;
	  REAL factotsa[NGRP];
	  for(igrp=0;igrp<NGRP;igrp++)
	      {
#ifdef OTSA
		factotsa[igrp]=0;
		alpha=alphab; // recombination is limited to non ground state levels
#else
		factotsa[igrp]=(igrp==0);
#endif

		ai_tmp1 = alphai[igrp];
		if(chemonly){
		  et[igrp]=egyloc[idloc+igrp*BLOCKCOOL];
		}
		else{
		  et[igrp]=((alpha-alphab)*x0[idloc]*x0[idloc]*nH[idloc]*nH[idloc]*dtcool*factotsa[igrp]+egyloc[idloc+igrp*BLOCKCOOL]+srcloc[idloc+igrp*BLOCKCOOL]*dtcool*factgrp[igrp])/(1.+dtcool*(ai_tmp1*(1.-x0[idloc])*nH[idloc]));
		}

		if((et[igrp]<0)||(isnan(et[igrp]))){
		  test=1;
		  //printf("eint=%e nH=%e x0=%e T=%e N=%e\n",eint[idloc],nH[idloc],x0[idloc],tloc,et[0]);
		}
		p[igrp]=(1.+(alphai[igrp]*nH[idloc]*(1-x0[idloc]))*dtcool);
	      }

	  ai_tmp1=0.;


	  if(test)
	    {
	      fudgecool=fudgecool/10.;
	      continue;
	    }

	  // IONISATION
#ifndef S_X
#ifdef SEMI_IMPLICIT
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += alphai[igrp]*et[igrp]*(!chemonly);}
#else
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += alphai[igrp]*egyloc[idloc+igrp*BLOCKCOOL]*(!chemonly);}
#endif
#else
	  N2[0]=1.0;
	  REAL pp=(1.-POW(x0[idloc],0.4092));
	  if(pp<0.) pp=0.;

	  for(igrp=1;igrp<NGRP;igrp++){
	    N2[igrp]=1.0+0.3908*POW(pp,1.7592)*E0overI[igrp];
	    if(N2[igrp]<1.0) N2[igrp]=1.0;
	  }
#ifdef SEMI_IMPLICIT
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += alphai[igrp]*et[igrp]*N2[igrp]*(!chemonly);}
#else
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += alphai[igrp]*egyloc[idloc+igrp*BLOCKCOOL]*N2[igrp]*(!chemonly);}
#endif
#endif

	  xt=1.-(alpha*x0[idloc]*x0[idloc]*nH[idloc]*dtcool+(1. -x0[idloc]))/(1.+dtcool*(beta*x0[idloc]*nH[idloc]+ai_tmp1));
	  ai_tmp1=0.;


	  if(((xt>1.)||(xt<0.))||(isnan(xt)))
 	    {
	      //printf("XION ERR eintt=%e xt=%e et=%e\n",eintt,xt,et[0]);
	      fudgecool/=10.;
	      continue;
	    }

#ifdef SEMI_IMPLICIT
	  dcuCompCooling(tloc,xt,nH[idloc],&Cool,&tcool1,aexp,CLUMPF2);
#else
	  dcuCompCooling(tloc,x0[idloc],nH[idloc],&Cool,&tcool1,aexp,CLUMPF2);
#endif

#ifdef COOLING
	  // HEATING + COOLING

	  int compcool=1; // do we need to compute the cooling ?

#ifdef SCHAYE
	  if((nH[idloc]>1e6)&&(R.nh>(param->stars->overdensity_cond*navg))){
	    REAL tlocs;
	    tlocs=eintt/(1.5*nH[idloc]*KBOLTZ*(1.+xt));
	    if(tlocs<1e5){
	      eintt=(1.08e9*KBOLTZ)*POW(nH[idloc]/1e5,4./3.)/(GAMMA-1)/FSCHAYE; // polytropic EOS
	      compcool=0.; // cancel cooling calculation
	      fudgecool=FMIN(fudgecool*1.5,param->fudgecool);
	    }
	  }
#endif // SCHAYE

	  if(compcool){
	    REAL SN = 0;
#ifdef SUPERNOVAE
	    SN 	 = R.snfb;
	    if (R.snfb) Cool = 0; // Stop the cooling if supernovae
	    if (R.snfb) printf("dE\t%e\tE0\t%e\tdtcool\t%e\t",R.snfb*dtcool,eintt, dtcool);
#endif

#ifndef S_X
#ifdef SEMI_IMPLICIT
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += et[igrp]*(alphae[igrp]*hnu[igrp]-(alphai[igrp]*hnu0))*(!chemonly);}
	  eintt=(eint[idloc]+ dtcool*(nH[idloc]*(1.-xt)*(ai_tmp1)-Cool+SN));
// 	  if (R.snfb) printf("E0\t%e\n",eintt);
#else
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += egyloc[idloc+igrp*BLOCKCOOL]*(alphae[igrp]*hnu[igrp]-(alphai[igrp]*hnu0))*(!chemonly);}
	  eintt=(eint[idloc]+dtcool*(nH[idloc]*(1.-x0[idloc])*(ai_tmp1)-Cool+SN));
#endif //SEMI


#else
	  //===================================== X RAYS ==============================
	  REAL pp2;
	  F2[0]=1.0;

	  //if(eint[idloc]!=E0) printf("7!\n");

#ifdef SEMI_IMPLICIT
	  pp2=1.0-POW(xt,0.2663);
#else
	  pp2=1.0-POW(x0[idloc],0.2663);
#endif
	  if(pp2<0.) pp2=0.;
	  for(igrp=1;igrp<NGRP;igrp++){
	    F2[igrp]=1.0;
	    F2[igrp]=0.9971*(1.0-POW(pp2,1.3163));

	    if(F2[igrp]>1.0) F2[igrp]=1.0;
	    if(F2[igrp]<0.0) F2[igrp]=0.0;
	  }

#ifdef SEMI_IMPLICIT
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += et[igrp]*(alphae[igrp]*hnu[igrp]-(alphai[igrp]*hnu0))*F2[igrp]*(!chemonly);}
	  eintt=(eint[idloc]+dtcool*(nH[idloc]*(1.-xt)*(ai_tmp1)-Cool+SN));
#else
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += egyloc[idloc+igrp*BLOCKCOOL]*(alphae[igrp]*hnu[igrp]-(alphai[igrp]*hnu0))*F2[igrp]*(!chemonly);}
	  eintt=(eint[idloc]+dtcool*(nH[idloc]*(1.-x0[idloc])*(ai_tmp1)-Cool+SN));
#endif
	  //================================================================================
#endif //S_X

	  if(eintt<0.)
 	    {
	      //printf("E NEG eintt=%e xt=%e et=%e\n",eintt,xt,et[0]);
	      fudgecool=fudgecool/10.;
	      continue;
	    }

	  if(FABS(eintt-eint[idloc])>FRAC_VAR*eint[idloc])
	    {
	      //	      if(srcloc[idloc]==0.){
	      //printf("DELTA E eintt=%e xt=%e et=%e\n",eintt,xt,et[0]);
	      fudgecool=fudgecool/10.;

	      continue;
	      //}
	    }
  	  else{
 	    fudgecool=FMIN(fudgecool*1.5,param->fudgecool);
	  }

	  ai_tmp1=0;


	  eintt=FMAX(emin,eintt);
 	  }

#else
	  eintt=eint[idloc];
#endif

	  // inner update
	  REAL aold=aexp;
#ifdef TESTCOSMO
	  da=hubblet*dtcool*aexp;
	  aexp+=da;
#endif

	  for(igrp =0;igrp<NGRP;igrp++)
	    {
	      egyloc[idloc+igrp*BLOCKCOOL]=et[igrp]*POW(aold/aexp,3);
	      if(!chemonly){
		floc[0+idloc3+igrp*BLOCKCOOL*3]=floc[0+idloc3+igrp*BLOCKCOOL*3]/p[igrp]*POW(aold/aexp,4);
		floc[1+idloc3+igrp*BLOCKCOOL*3]=floc[1+idloc3+igrp*BLOCKCOOL*3]/p[igrp]*POW(aold/aexp,4);
		floc[2+idloc3+igrp*BLOCKCOOL*3]=floc[2+idloc3+igrp*BLOCKCOOL*3]/p[igrp]*POW(aold/aexp,4);
	      }
	    }

	  x0[idloc]=xt;
	  //printf("xt=%e\n",xt);
#ifdef COOLING
	  eint[idloc]=eintt*POW(aold/aexp,5);
#endif

	  currentcool_t+=dtcool;
	  fudgecool=param->fudgecool;
	  nitcool++;
	  if((nitcool==ncvgcool)&&(ncvgcool!=0)) break;
	}

      /// ====================== End of the cooling loop

      //aexp=aexporg;
      // FIlling the rad structure to send it back

      if(!chemonly){
	for(igrp=0;igrp<NGRP;igrp++)
	  {
	    R.e[igrp]=FMAX(egyloc[idloc+igrp*BLOCKCOOL]*POW(aexp,3),EMIN*factgrp[igrp])/param->unit.unit_N;
	    R.fx[igrp]=floc[0+idloc3+igrp*BLOCKCOOL*3]*POW(aexp,4)/param->unit.unit_l*param->unit.unit_t/param->unit.unit_N;
	    R.fy[igrp]=floc[1+idloc3+igrp*BLOCKCOOL*3]*POW(aexp,4)/param->unit.unit_l*param->unit.unit_t/param->unit.unit_N;
	    R.fz[igrp]=floc[2+idloc3+igrp*BLOCKCOOL*3]*POW(aexp,4)/param->unit.unit_l*param->unit.unit_t/param->unit.unit_N;
	  }
      }

      R.nhplus=x0[idloc]*R.nh;
      R.eint=eint[idloc]*POW(aexp,5)/param->unit.unit_n/param->unit.unit_d/POW(param->unit.unit_v,2);
      dE2T(&R,aexp,param);
      memcpy(&stencil[i].New.cell[icell].rfieldnew,&R,sizeof(struct Rtype));

    }
  }
}


#if 0
__global__ void dchemrad(struct RGRID *stencil, int nread, int stride, struct CPUINFO *cpu, REAL dxcur, REAL dtnew, struct RUNPARAMS *param, REAL aexporg, int chemonly)
{
  int i,icell,igrp;
  //int idloc;
  int nitcool=0;

  REAL hnu0=13.6*1.6022e-19,
    Cool,
    tcool,
    dtcool,
    tcool1,
    currentcool_t=0.,
    alpha,
    alphab,
    beta,
    tloc,
    xt,
    eintt,
    ai_tmp1=0.,
    et[NGRP],
    p[NGRP];

  REAL aexp;
  REAL ebkg[NGRP];
  REAL z=1./aexporg-1.;

  REAL c=param->clightorg*LIGHT_SPEED_IN_M_PER_S; 			// switch back to physical velocity m/s

  REAL hnu[NGRP];
  REAL alphae[NGRP];
  REAL alphai[NGRP];
  REAL factgrp[NGRP];

  for(igrp=0;igrp<NGRP;igrp++) {
    hnu[igrp]=param->atomic.hnu[igrp];
    alphae[igrp]=param->atomic.alphae[igrp];
    alphai[igrp]=param->atomic.alphai[igrp];
    factgrp[igrp]=param->atomic.factgrp[igrp];
  }


#ifdef S_X
  REAL E0overI[NGRP];
  REAL N2[NGRP];
  REAL F2[NGRP];
#endif


#define BLOCKCOOL 1 // KEPT FROM CUDATON FOR SIMPLICITY
#define idloc 0 // KEPT FROM CUDATON FOR SIMPLICITY
#define idloc3 0 // KEPT FROM CUDATON FOR SIMPLICITY

  REAL
    egyloc[BLOCKCOOL*NGRP],
    floc[3*BLOCKCOOL*NGRP],
    x0[BLOCKCOOL],
    nH[BLOCKCOOL],
    eint[BLOCKCOOL],
    srcloc[BLOCKCOOL];
  
  
  REAL dt=dtnew*param->unit.unit_t*POW(aexporg,2);

  REAL emin;
  struct Rtype R;
  REAL fudgecool=param->fudgecool;
  int ncvgcool=param->ncvgcool;
  REAL E0;
#ifdef SCHAYE
  REAL navg=(param->cosmo->ob/param->cosmo->om)/(PROTON_MASS*MOLECULAR_MU)*param->unit.unit_d;
#endif
  REAL xorg;

  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
  i=bx*blockDim.x+tx;
  
  if(i<nread){
    for(icell=0;icell<8;icell++){ // we scan the cells

     
      if(stencil[i].oct[6].cell[icell].split) continue; // we dont treat split cells
      memcpy(&R,&stencil[i].New.cell[icell].rfieldnew,sizeof(struct Rtype));// We get the local physical quantities after transport update
      
#ifdef HOMOSOURCE
      // we override the value with the homogeneous source density
      R.src=param->bkg;
#endif

      // switch to physical units, chemistry remains unchanged with and without cosmo
      for (igrp=0;igrp<NGRP;igrp++)
	{			
	  egyloc[idloc+igrp*BLOCKCOOL]   =R.e[igrp]/(aexporg*aexporg*aexporg)*param->unit.unit_N;//+ebkg[igrp]; 
	  floc[0+idloc3+igrp*BLOCKCOOL*3]=R.fx[igrp]/POW(aexporg,4)*param->unit.unit_l/param->unit.unit_t*param->unit.unit_N;
	  floc[1+idloc3+igrp*BLOCKCOOL*3]=R.fy[igrp]/POW(aexporg,4)*param->unit.unit_l/param->unit.unit_t*param->unit.unit_N;
	  floc[2+idloc3+igrp*BLOCKCOOL*3]=R.fz[igrp]/POW(aexporg,4)*param->unit.unit_l/param->unit.unit_t*param->unit.unit_N;
	}


      x0[idloc]=R.nhplus/R.nh;
      xorg= x0[idloc];
      nH[idloc]=R.nh/(aexporg*aexporg*aexporg)*param->unit.unit_N;

      eint[idloc]=R.eint/POW(aexporg,5)*param->unit.unit_n*param->unit.unit_d*POW(param->unit.unit_v,2);
      emin=PMIN/(GAMMA-1.)/POW(aexporg,5)*param->unit.unit_n*param->unit.unit_d*POW(param->unit.unit_v,2); // physical minimal pressure
      //srcloc[idloc]=(R.src*param->unit.unit_N/param->unit.unit_t/(aexporg*aexporg)+ebkg[0])/POW(aexporg,3); 
      for (igrp=0;igrp<NGRP;igrp++){
      srcloc[idloc+igrp*BLOCKCOOL]=(R.src[igrp]*param->unit.unit_N/param->unit.unit_t/(aexporg*aexporg))/POW(aexporg,3); //phot/s/dv (physique)
      }

      /// BELOW THE FULL EXPRESSION OF E in natural units
      //emin=PMIN/(GAMMA-1.)/POW(aexporg,5)/POW(param->unit.unit_l,3)*param->unit.unit_n*param->unit.unit_d*POW(param->unit.unit_v,2); // physical minimal pressure


      /* if(((isnan(eint[idloc]))||(isnan(x0[idloc])))||(eint[idloc]==0.)){ */
      /* 	printf("start with nans or ZErO egy %e\n",eint[idloc]); */
      /* 	abort(); */
      /* } */

      // at this stage we are ready to do the calculations

      // DEALING WITH CLUMPING ----------------------
#ifdef WCLUMP
      REAL CLUMPF2=FMIN(FMAX(POW(nH[idloc]/6.,0.7),1.),40.);
      REAL CLUMPI=1.;
#else
      REAL CLUMPF2=1.;
      REAL CLUMPI=1.;
#endif


      for(igrp=0;igrp<NGRP;igrp++)
	{
	  alphai[igrp] *= CLUMPI;
	  alphae[igrp] *= CLUMPI;
	}

      // -------------------------------------------------


      // local cooling loop -------------------------------
      aexp=aexporg;
      fudgecool=param->fudgecool;
      currentcool_t=0.;
      nitcool=0.;
      REAL da;
      //printf("fudge=%e ncv=%d currentcool_t=%e dt=%e\n",fudgecool,ncvgcool,currentcool_t,dt);
#if 1
      while(currentcool_t<dt)
	{

	  // Cosmological Adiabatic expansion effects ==============
#ifdef TESTCOSMO
	  REAL hubblet=param->cosmo->H0*SQRT(param->cosmo->om/aexp+param->cosmo->ov*(aexp*aexp))/aexp*(1e3/(1e6*PARSEC)); // s-1 // SOMETHING TO CHECK HERE
#else
	  REAL hubblet=0.;
#endif
	  tloc=eint[idloc]/(1.5*nH[idloc]*KBOLTZ*(1.+x0[idloc]));

	  //== Getting a timestep
	  dcuCompCooling(tloc,x0[idloc],nH[idloc],&Cool,&tcool1,aexp,CLUMPF2);
	  ai_tmp1=0.;

	  for (igrp=0;igrp<NGRP;igrp++) ai_tmp1 += ((alphae[igrp])*hnu[igrp]-(alphai[igrp])*hnu0)*egyloc[idloc+igrp*BLOCKCOOL];
	  
	  tcool=FABS(eint[idloc]/(nH[idloc]*(1.0-x0[idloc])*ai_tmp1-Cool));
	  ai_tmp1=0.;
	  dtcool=FMIN(fudgecool*tcool,dt-currentcool_t);
	  
	  alpha=dcucompute_alpha_a(tloc,1.,1.)*CLUMPF2;
	  alphab=dcucompute_alpha_b(tloc,1.,1.)*CLUMPF2;
	  beta=dcucompute_beta(tloc,1.,1.)*CLUMPF2;
      
	  //== Update
	  
	  // ABSORPTION
	  int test = 0;
	  REAL factotsa[NGRP];
	  for(igrp=0;igrp<NGRP;igrp++)
	    {
#ifdef OTSA
	      factotsa[igrp]=0;
	      alpha=alphab; // recombination is limited to non ground state levels
#else
	      factotsa[igrp]=(igrp==0);
#endif

	      ai_tmp1 = alphai[igrp];
	      if(chemonly){
		et[igrp]=egyloc[idloc+igrp*BLOCKCOOL];
		}
	      else{
		et[igrp]=((alpha-alphab)*x0[idloc]*x0[idloc]*nH[idloc]*nH[idloc]*dtcool*factotsa[igrp]+egyloc[idloc+igrp*BLOCKCOOL]+srcloc[idloc+igrp*BLOCKCOOL]*dtcool*factgrp[igrp])/(1.+dtcool*(ai_tmp1*(1.-x0[idloc])*nH[idloc]));
	      }

	      if((et[igrp]<0)||(isnan(et[igrp]))){
		test=1;
	      }
	      p[igrp]=(1.+(alphai[igrp]*nH[idloc]*(1-x0[idloc]))*dtcool);
	    }
	  
	  ai_tmp1=0.;

	  
	  if(test) 
	    {
	      fudgecool=fudgecool/10.; 
	      continue;	
	    } 

	  
	  // IONISATION
#ifndef S_X
#ifdef SEMI_IMPLICIT
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += alphai[igrp]*et[igrp]*(!chemonly);}
#else
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += alphai[igrp]*egyloc[idloc+igrp*BLOCKCOOL]*(!chemonly);}
#endif
#else
	  N2[0]=1.0;
	  REAL pp=(1.-POW(x0[idloc],0.4092)); 
	  if(pp<0.) pp=0.; 

	  for(igrp=1;igrp<NGRP;igrp++){
	    N2[igrp]=1.0+0.3908*POW(pp,1.7592)*E0overI[igrp]; 
	    if(N2[igrp]<1.0) N2[igrp]=1.0; 
	  }
#ifdef SEMI_IMPLICIT
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += alphai[igrp]*et[igrp]*N2[igrp]*(!chemonly);}
#else
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += alphai[igrp]*egyloc[idloc+igrp*BLOCKCOOL]*N2[igrp]*(!chemonly);}
#endif
#endif
	  
	  xt=1.-(alpha*x0[idloc]*x0[idloc]*nH[idloc]*dtcool+(1. -x0[idloc]))/(1.+dtcool*(beta*x0[idloc]*nH[idloc]+ai_tmp1));
	  ai_tmp1=0.;


	  if(((xt>1.)||(xt<0.))||(isnan(xt))) 
 	    {
	      fudgecool/=10.; 
	      continue;	
	    } 

#ifdef SEMI_IMPLICIT
	  dcuCompCooling(tloc,xt,nH[idloc],&Cool,&tcool1,aexp,CLUMPF2);
#else
	  dcuCompCooling(tloc,x0[idloc],nH[idloc],&Cool,&tcool1,aexp,CLUMPF2);
#endif

#ifdef COOLING
	  // HEATING
	  
	  int compcool=1; // do we need to compute the cooling ?

#ifdef SCHAYE
	  if((nH[idloc]>1e5)&&(R.nh>(57.7*navg))){
	    REAL tlocs;
	    tlocs=eintt/(1.5*nH[idloc]*KBOLTZ*(1.+xt));
	    if(tlocs<1e5){
	      eintt=(1.08e9*KBOLTZ)*POW(nH[idloc]/1e5,4./3.)/(GAMMA-1); // polytropic EOS
	      compcool=0.; // cancel cooling calculation
	      fudgecool=FMIN(fudgecool*1.5,param->fudgecool);
	    }
	  }
#endif

	  if(compcool){ 
	  REAL SN=0.;

#ifndef S_X
#ifdef SEMI_IMPLICIT
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += et[igrp]*(alphae[igrp]*hnu[igrp]-(alphai[igrp]*hnu0))*(!chemonly);}
	  eintt=(eint[idloc]+dtcool*(nH[idloc]*(1.-xt)*(ai_tmp1)-Cool));
#else
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += egyloc[idloc+igrp*BLOCKCOOL]*(alphae[igrp]*hnu[igrp]-(alphai[igrp]*hnu0))*(!chemonly);}
	  eintt=(eint[idloc]+dtcool*(nH[idloc]*(1.-x0[idloc])*(ai_tmp1)-Cool));
#endif
#else
	  //===================================== X RAYS ==============================
	  REAL pp2;
	  F2[0]=1.0;
#ifdef SEMI_IMPLICIT
	  pp2=1.0-POW(xt,0.2663); 
#else
	  pp2=1.0-POW(x0[idloc],0.2663); 
#endif
	  if(pp2<0.) pp2=0.; 
	  for(igrp=1;igrp<NGRP;igrp++){
	    F2[igrp]=1.0;
	    F2[igrp]=0.9971*(1.0-POW(pp2,1.3163)); 
	  
	    if(F2[igrp]>1.0) F2[igrp]=1.0; 
	    if(F2[igrp]<0.0) F2[igrp]=0.0; 
	  }
#ifdef SEMI_IMPLICIT
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += et[igrp]*(alphae[igrp]*hnu[igrp]-(alphai[igrp]*hnu0))*F2[igrp]*(!chemonly);}
	  eintt=(eint[idloc]+dtcool*(nH[idloc]*(1.-xt)*(ai_tmp1)-Cool));
#else
	  for(igrp=0;igrp<NGRP;igrp++) {ai_tmp1 += egyloc[idloc+igrp*BLOCKCOOL]*(alphae[igrp]*hnu[igrp]-(alphai[igrp]*hnu0))*F2[igrp]*(!chemonly);}
	  eintt=(eint[idloc]+dtcool*(nH[idloc]*(1.-x0[idloc])*(ai_tmp1)-Cool));
#endif
#endif

	  if(eintt<0.)
 	    {
	      fudgecool=fudgecool/10.;

	      continue;
	    }

	  if(FABS(eintt-eint[idloc])>FRAC_VAR*eint[idloc])
	    {
	      fudgecool=fudgecool/10.;
	      continue;
	    }
  	  else{

 	    fudgecool=FMIN(fudgecool*1.5,param->fudgecool);
	  }

	  ai_tmp1=0;
	  
	  eintt=FMAX(emin,eintt);
	  }
#else
	  eintt=eint[idloc];
#endif

	  // inner update
	  REAL aold=aexp;
#ifdef TESTCOSMO
	  REAL da=hubblet*dtcool*aexp;
	  aexp+=da;
#endif

	  for(igrp =0;igrp<NGRP;igrp++)
	    {
	      egyloc[idloc+igrp*BLOCKCOOL]=et[igrp]*POW(aold/aexp,3);
	      if(!chemonly){
		floc[0+idloc3+igrp*BLOCKCOOL*3]=floc[0+idloc3+igrp*BLOCKCOOL*3]/p[igrp]*POW(aold/aexp,4);
		floc[1+idloc3+igrp*BLOCKCOOL*3]=floc[1+idloc3+igrp*BLOCKCOOL*3]/p[igrp]*POW(aold/aexp,4);
		floc[2+idloc3+igrp*BLOCKCOOL*3]=floc[2+idloc3+igrp*BLOCKCOOL*3]/p[igrp]*POW(aold/aexp,4);
	      }	
	    }
	  
	  x0[idloc]=xt;
#ifdef COOLING
	  eint[idloc]=eintt*POW(aold/aexp,5);
#endif
	  currentcool_t+=dtcool;
	  fudgecool=param->fudgecool;
	  nitcool++;
	  if((nitcool==ncvgcool)&&(ncvgcool!=0)) break;
	}
#endif
      // ====================== End of the cooling loop

      // FIlling the rad structure to send it back
       if(!chemonly){
	 for(igrp=0;igrp<NGRP;igrp++)
	   {
	     R.e[igrp]=FMAX(egyloc[idloc+igrp*BLOCKCOOL]*aexp*aexp*aexp,EMIN*factgrp[igrp])/param->unit.unit_N;
	     R.fx[igrp]=floc[0+idloc3+igrp*BLOCKCOOL*3]*POW(aexp,4)/param->unit.unit_l*param->unit.unit_t/param->unit.unit_N;
	     R.fy[igrp]=floc[1+idloc3+igrp*BLOCKCOOL*3]*POW(aexp,4)/param->unit.unit_l*param->unit.unit_t/param->unit.unit_N;
	     R.fz[igrp]=floc[2+idloc3+igrp*BLOCKCOOL*3]*POW(aexp,4)/param->unit.unit_l*param->unit.unit_t/param->unit.unit_N;
	   }
       }
      R.nhplus=x0[idloc]*R.nh;
      R.eint=eint[idloc]*POW(aexp,5)/param->unit.unit_n/param->unit.unit_d/POW(param->unit.unit_v,2);
      dE2T(&R,aexp,param);
      memcpy(&stencil[i].New.cell[icell].rfieldnew,&R,sizeof(struct Rtype));

    }
  }

}
#endif
#endif
#endif
