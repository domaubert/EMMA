
#include <hip/hip_runtime.h>

#ifdef WHYDRO2

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "prototypes.h"
#include <mpi.h>
//#include <cudpp.h>
#include "gpu_type.h"

#define NITERMAX 10
#define ERRTOL 1e-10
#define DEFDENSG 10.

extern "C" struct OCT *gatherstencil(struct OCT *octstart, struct HGRID *stencil, int stride, struct CPUINFO *cpu, int *nread);
extern "C" struct OCT *scatterstencil(struct OCT *octstart, struct HGRID *stencil, int stride, struct CPUINFO *cpu, REAL dxcur, REAL dtnew);
extern "C" void create_hydstencil_GPU(struct CPUINFO *cpu, int stride);
extern "C" int advancehydroGPU(struct OCT **firstoct, int level, struct CPUINFO *cpu, struct HGRID *stencil, int stride, REAL dxcur, REAL dtnew);
extern "C" void create_pinned_stencil(struct HGRID **stencil, int stride);
extern "C" void destroy_hydstencil_GPU(struct CPUINFO *cpu, int stride);
extern "C" void destroy_pinned_stencil(struct HGRID **stencil, int stride);

// ===================================================================
void create_hydstencil_GPU(struct CPUINFO *cpu, int stride){
  cudaMalloc((void **)&(cpu->hyd_stencil),sizeof(struct HGRID)*stride);
}

// ===================================================================
void create_pinned_stencil(struct HGRID **stencil, int stride){
  cudaMallocHost( (void**)stencil, sizeof(struct HGRID)*stride );
  CUDA_CHECK_ERROR("GPU hydro alloc");
}

// ===================================================================
void destroy_hydstencil_GPU(struct CPUINFO *cpu, int stride){
  cudaFree(cpu->hyd_stencil);
}

// ===================================================================
void destroy_pinned_stencil(struct HGRID **stencil, int stride){
  cudaFreeHost(stencil);
}



__device__ void dinitUtype(struct Utype* u){
  u->d=0;
  u->du=0;
  u->dv=0;
  u->dw=0;
  u->E=0;
  u->eint=0;

#ifdef WRADHYD
  u->dX=0;
#ifdef HELIUM
  u->dXHE=0;
  u->dXXHE=0;
#endif // HELIUM
#endif // WRADHYD
}
// ==============================================================================================================

__device__ void dgetE(struct Wtype *W){
  W->E=W->p/(GAMMA-1.)+0.5*W->d*(W->u*W->u+W->v*W->v+W->w*W->w);
}

// =======================================================

__device__ void getcellnei_gpu_hydro(int cindex, int *neip, int *cell)
{
  switch(cindex){
  case 0:
    neip[0]=0;cell[0]=1;
    neip[1]=6;cell[1]=1;
    neip[2]=2;cell[2]=2;
    neip[3]=6;cell[3]=2;
    neip[4]=4;cell[4]=4;
    neip[5]=6;cell[5]=4;
    break;
  case 1:
    neip[0]=6;cell[0]=0;
    neip[1]=1;cell[1]=0;
    neip[2]=2;cell[2]=3;
    neip[3]=6;cell[3]=3;
    neip[4]=4;cell[4]=5;
    neip[5]=6;cell[5]=5;
    break;
  case 2:
    neip[0]=0;cell[0]=3;
    neip[1]=6;cell[1]=3;
    neip[2]=6;cell[2]=0;
    neip[3]=3;cell[3]=0;
    neip[4]=4;cell[4]=6;
    neip[5]=6;cell[5]=6;
    break;
  case 3:
    neip[0]=6;cell[0]=2;
    neip[1]=1;cell[1]=2;
    neip[2]=6;cell[2]=1;
    neip[3]=3;cell[3]=1;
    neip[4]=4;cell[4]=7;
    neip[5]=6;cell[5]=7;
    break;
  case 4:
    neip[0]=0;cell[0]=5;
    neip[1]=6;cell[1]=5;
    neip[2]=2;cell[2]=6;
    neip[3]=6;cell[3]=6;
    neip[4]=6;cell[4]=0;
    neip[5]=5;cell[5]=0;
    break;
  case 5:
    neip[0]=6;cell[0]=4;
    neip[1]=1;cell[1]=4;
    neip[2]=2;cell[2]=7;
    neip[3]=6;cell[3]=7;
    neip[4]=6;cell[4]=1;
    neip[5]=5;cell[5]=1;
    break;
  case 6:
    neip[0]=0;cell[0]=7;
    neip[1]=6;cell[1]=7;
    neip[2]=6;cell[2]=4;
    neip[3]=3;cell[3]=4;
    neip[4]=6;cell[4]=2;
    neip[5]=5;cell[5]=2;
    break;
  case 7:
    neip[0]=6;cell[0]=6;
    neip[1]=1;cell[1]=6;
    neip[2]=6;cell[2]=5;
    neip[3]=3;cell[3]=5;
    neip[4]=6;cell[4]=3;
    neip[5]=5;cell[5]=3;
    break;
  }

}

// ==================== converts U -> W

__device__ void dU2W(struct Utype *U, struct Wtype *W)
{

  REAL dloc=(U->d==0.?DEFDENSG:U->d);

  W->d=U->d;
  W->u=U->du/dloc;
  W->v=U->dv/dloc;
  W->w=U->dw/dloc;

#ifdef DUAL_E
  W->p=U->eint*(GAMMA-1.);
#else
  W->p=(GAMMA-1.)*(U->E-((U->du)*(U->du)+(U->dv)*(U->dv)+(U->dw)*(U->dw))/(dloc)*0.5);
#endif

  W->E=U->E;

#ifdef WRADHYD
  W->dX=U->dX;

#ifdef HELIUM
  W->dXHE=U->dXHE;
  W->dXXHE=U->dXXHE;
#endif // HELIUM
#endif // WRADHYD

  W->a=SQRT(GAMMA*W->p/dloc);
}


// ==================== converts W -> U
__device__ void dW2U(struct Wtype *W, struct Utype *U)
{
  U->d=W->d;
  U->du=W->d*W->u;
  U->dv=W->d*W->v;
  U->dw=W->d*W->w;

  U->eint=W->p/(GAMMA-1.);
  U->E=W->E;

#ifdef WRADHYD
  U->dX=W->dX;
#ifdef HELIUM
  U->dXHE=W->dXHE;
  U->dXXHE=W->dXXHE;
#endif
#endif

}



// ---------------------------------------------------------------
__device__ void dgetflux_X(struct Utype *U, REAL *f)
{
  f[0]=U->du;
  f[1]=0.5*(3.-GAMMA)*U->du*U->du/U->d+(GAMMA-1.)*U->E-0.5*(GAMMA-1.)*(U->dv*U->dv+U->dw*U->dw)/U->d;
  f[2]=U->du*U->dv/U->d;
  f[3]=U->du*U->dw/U->d;
  f[4]=GAMMA*U->du/U->d*U->E-0.5*(GAMMA-1.)*U->du/(U->d*U->d)*(U->du*U->du+U->dv*U->dv+U->dw*U->dw);
#ifdef WRADHYD
  f[6]=U->du*U->dX/U->d;
#ifdef HELIUM
  f[7]=U->du*U->dXHE/U->d;
  f[8]=U->du*U->dXXHE/U->d;
#endif

#endif
}

// ---------------------------------------------------------------

__device__ void dgetflux_Y(struct Utype *U, REAL *f)
{
  f[0]=U->dv;
  f[1]=U->dv*U->du/U->d;
  f[2]=0.5*(3.-GAMMA)*U->dv*U->dv/U->d+(GAMMA-1.)*U->E-0.5*(GAMMA-1.)*(U->du*U->du+U->dw*U->dw)/U->d;
  f[3]=U->dv*U->dw/U->d;
  f[4]=GAMMA*U->dv/U->d*U->E-0.5*(GAMMA-1.)*U->dv/(U->d*U->d)*(U->du*U->du+U->dv*U->dv+U->dw*U->dw);
#ifdef WRADHYD
  f[6]=U->dv*U->dX/U->d;
#ifdef HELIUM
  f[7]=U->dv*U->dXHE/U->d;
  f[8]=U->dv*U->dXXHE/U->d;
#endif

#endif
}

// ---------------------------------------------------------------

__device__ void dgetflux_Z(struct Utype *U, REAL *f)
{
  f[0]=U->dw;
  f[1]=U->dw*U->du/U->d;
  f[2]=U->dw*U->dv/U->d;
  f[3]=0.5*(3.-GAMMA)*U->dw*U->dw/U->d+(GAMMA-1.)*U->E-0.5*(GAMMA-1.)*(U->du*U->du+U->dv*U->dv)/U->d;
  f[4]=GAMMA*U->dw/U->d*U->E-0.5*(GAMMA-1.)*U->dw/(U->d*U->d)*(U->du*U->du+U->dv*U->dv+U->dw*U->dw);
#ifdef WRADHYD
  f[6]=U->dw*U->dX/U->d;
#ifdef HELIUM
  f[7]=U->dw*U->dXHE/U->d;
  f[8]=U->dw*U->dXXHE/U->d;
#endif

#endif
}



// ================== performs the difference between two Us

__device__ void ddiffU(struct Utype *U2, struct Utype *U1, struct Utype *UR){
  
  UR->d =U2->d - U1->d;
  UR->du=U2->du- U1->du;
  UR->dv=U2->dv- U1->dv;
  UR->dw=U2->dw- U1->dw;
  UR->E =U2->E - U1->E;
  UR->eint=U2->eint-U1->eint;
}

// ================== performs the difference between two Ws

__device__ void ddiffW(struct Wtype *W2, struct Wtype *W1, struct Wtype *WR){

  WR->d=W2->d- W1->d;
  WR->u=W2->u- W1->u;
  WR->v=W2->v- W1->v;
  WR->w=W2->w- W1->w;
  WR->p=W2->p- W1->p;
#ifdef WRADHYD
  WR->dX=W2->dX- W1->dX;
#ifdef HELIUM
  WR->dXHE=W2->dXHE- W1->dXHE;
  WR->dXXHE=W2->dXXHE- W1->dXXHE;
#endif
#endif
}




// ================= minmod
__device__ void dminmod(struct Utype *Um, struct Utype *Up, struct Utype *Ur){

  REAL beta=1.; // 1. for MINBEE 2. for SUPERBEE
  // FLUX LIMITER

  if(Up->d>0){
    Ur->d=FMAX(FMAX(0.,FMIN(beta*Um->d,Up->d)),FMIN(Um->d,beta*Up->d));
  }
  else{
    Ur->d=FMIN(FMIN(0.,FMAX(beta*Um->d,Up->d)),FMAX(Um->d,beta*Up->d));
  }


  if(Up->du>0){
    Ur->du=FMAX(FMAX(0.,FMIN(beta*Um->du,Up->du)),FMIN(Um->du,beta*Up->du));
  }
  else{
    Ur->du=FMIN(FMIN(0.,FMAX(beta*Um->du,Up->du)),FMAX(Um->du,beta*Up->du));
  }


  if(Up->dv>0){
    Ur->dv=FMAX(FMAX(0.,FMIN(beta*Um->dv,Up->dv)),FMIN(Um->dv,beta*Up->dv));
  }
  else{
    Ur->dv=FMIN(FMIN(0.,FMAX(beta*Um->dv,Up->dv)),FMAX(Um->dv,beta*Up->dv));
  }


  if(Up->dw>0){
    Ur->dw=FMAX(FMAX(0.,FMIN(beta*Um->dw,Up->dw)),FMIN(Um->dw,beta*Up->dw));
  }
  else{
    Ur->dw=FMIN(FMIN(0.,FMAX(beta*Um->dw,Up->dw)),FMAX(Um->dw,beta*Up->dw));
  }


  if(Up->E>0){
    Ur->E=FMAX(FMAX(0.,FMIN(beta*Um->E,Up->E)),FMIN(Um->E,beta*Up->E));
  }
  else{
    Ur->E=FMIN(FMIN(0.,FMAX(beta*Um->E,Up->E)),FMAX(Um->E,beta*Up->E));
  }


}

//===============================================
//===============================================
__device__ void dminmod_W(struct Wtype *Wm, struct Wtype *Wp, struct Wtype *Wr){

  REAL beta=1.; // 1. for MINBEE 2. for SUPERBEE
  // FLUX LIMITER

  if(Wp->d>0){
    Wr->d=FMAX(FMAX(0.,FMIN(beta*Wm->d,Wp->d)),FMIN(Wm->d,beta*Wp->d));
  }
  else{
    Wr->d=FMIN(FMIN(0.,FMAX(beta*Wm->d,Wp->d)),FMAX(Wm->d,beta*Wp->d));
  }

#ifdef WRADHYD
  if(Wp->dX>0){
    Wr->dX=FMAX(FMAX(0.,FMIN(beta*Wm->dX,Wp->dX)),FMIN(Wm->dX,beta*Wp->dX));
  }
  else{
    Wr->dX=FMIN(FMIN(0.,FMAX(beta*Wm->dX,Wp->dX)),FMAX(Wm->dX,beta*Wp->dX));
  }
#ifdef HELIUM
  if(Wp->dXHE>0){
    Wr->dXHE=FMAX(FMAX(0.,FMIN(beta*Wm->dXHE,Wp->dXHE)),FMIN(Wm->dXHE,beta*Wp->dXHE));
  }
  else{
    Wr->dXHE=FMIN(FMIN(0.,FMAX(beta*Wm->dXHE,Wp->dXHE)),FMAX(Wm->dXHE,beta*Wp->dXHE));
  }

  if(Wp->dXXHE>0){
    Wr->dXXHE=FMAX(FMAX(0.,FMIN(beta*Wm->dXXHE,Wp->dXXHE)),FMIN(Wm->dXXHE,beta*Wp->dXXHE));
  }
  else{
    Wr->dXXHE=FMIN(FMIN(0.,FMAX(beta*Wm->dXXHE,Wp->dXXHE)),FMAX(Wm->dXXHE,beta*Wp->dXXHE));
  }
#endif // HELIUM
#endif // WRADHYD

  if(Wp->u>0){
    Wr->u=FMAX(FMAX(0.,FMIN(beta*Wm->u,Wp->u)),FMIN(Wm->u,beta*Wp->u));
  }
  else{
    Wr->u=FMIN(FMIN(0.,FMAX(beta*Wm->u,Wp->u)),FMAX(Wm->u,beta*Wp->u));
  }


  if(Wp->v>0){
    Wr->v=FMAX(FMAX(0.,FMIN(beta*Wm->v,Wp->v)),FMIN(Wm->v,beta*Wp->v));
  }
  else{
    Wr->v=FMIN(FMIN(0.,FMAX(beta*Wm->v,Wp->v)),FMAX(Wm->v,beta*Wp->v));
  }


  if(Wp->w>0){
    Wr->w=FMAX(FMAX(0.,FMIN(beta*Wm->w,Wp->w)),FMIN(Wm->w,beta*Wp->w));
  }
  else{
    Wr->w=FMIN(FMIN(0.,FMAX(beta*Wm->w,Wp->w)),FMAX(Wm->w,beta*Wp->w));
  }


  if(Wp->p>0){
    Wr->p=FMAX(FMAX(0.,FMIN(beta*Wm->p,Wp->p)),FMIN(Wm->p,beta*Wp->p));
  }
  else{
    Wr->p=FMIN(FMIN(0.,FMAX(beta*Wm->p,Wp->p)),FMAX(Wm->p,beta*Wp->p));
  }


}



// ============= interp minmod ====================================================

__device__ void dinterpminmod(struct Utype *U0, struct Utype *Up, struct Utype *Dx, struct Utype *Dy, struct Utype *Dz,REAL dx,REAL dy,REAL dz){
  
  Up->d =U0->d  + dx*Dx->d  +dy*Dy->d  +dz*Dz->d;
  Up->du=U0->du + dx*Dx->du +dy*Dy->du +dz*Dz->du;
  Up->dv=U0->dv + dx*Dx->dv +dy*Dy->dv +dz*Dz->dv;
  Up->dw=U0->dw + dx*Dx->dw +dy*Dy->dw +dz*Dz->dw;
  Up->E =U0->E  + dx*Dx->E  +dy*Dy->E  +dz*Dz->E;
  Up->eint =U0->eint  + dx*Dx->eint  +dy*Dy->eint  +dz*Dz->eint;
}

// ============= interp minmod ====================================================

__device__ void dinterpminmod_W(struct Wtype *W0, struct Wtype *Wp, struct Wtype *Dx, struct Wtype *Dy, struct Wtype *Dz,REAL dx,REAL dy,REAL dz){
  
  Wp->d =W0->d +dx*Dx->d +dy*Dy->d +dz*Dz->d;
  Wp->u =W0->u +dx*Dx->u +dy*Dy->u +dz*Dz->u;
  Wp->v =W0->v +dx*Dx->v +dy*Dy->v +dz*Dz->v;
  Wp->w =W0->w +dx*Dx->w +dy*Dy->w +dz*Dz->w;
  Wp->p =W0->p +dx*Dx->p +dy*Dy->p +dz*Dz->p;
#ifdef WRADHYD
  Wp->dX =W0->dX +dx*Dx->dX +dy*Dy->dX +dz*Dz->dX;
#ifdef HELIUM
  Wp->dXHE =W0->dXHE +dx*Dx->dXHE +dy*Dy->dXHE +dz*Dz->dXHE;
  Wp->dXXHE =W0->dXXHE +dx*Dx->dXXHE +dy*Dy->dXXHE +dz*Dz->dXXHE;
#endif
#endif

}


__device__ void  dmatrix_jacobian(struct Wtype *W0, REAL dt,REAL dx,struct Wtype *Dx,struct Wtype *Dy,struct Wtype *Dz, struct Wtype *Wt){


  REAL M[25];
#ifdef HELIUM
  REAL W[8]={0.,0.,0.,0.,0.,0.,0.,0.};
#else
  REAL W[6]={0.,0.,0.,0.,0.,0.};
#endif // HELIUM
  REAL d[5];
  int i,j;
#ifdef WRADHYD
  REAL X;
#endif

  // =====  building the A matrix

  memset(M,0,25*sizeof(REAL));

  // diagonal elements
  for(i=0;i<5;i++) M[i+i*5]=W0->u;

  // off_diagonal elements
  M[0+1*5]=W0->d;

  M[4+1*5]=W0->d*W0->a*W0->a;

  M[1+4*5]=1./W0->d;


  // ===== First Product

  d[0]=Dx->d;
  d[1]=Dx->u;
  d[2]=Dx->v;
  d[3]=Dx->w;
  d[4]=Dx->p;

  for(j=0;j<5;j++){
    for(i=0;i<5;i++){
      W[i]+=M[i+j*5]*d[j];
    }
  }

#ifdef WRADHYD
  W[5]+=W0->u*Dx->dX+W0->dX*Dx->u;
#ifdef HELIUM
  W[6]+=W0->u*Dx->dXHE+W0->dXHE*Dx->u;
  W[7]+=W0->u*Dx->dXXHE+W0->dXXHE*Dx->u;
#endif // HELIUM
#endif // WRADHYD

  // =====  building the B matrix

  memset(M,0,25*sizeof(REAL));

  // diagonal elements
  for(i=0;i<5;i++) M[i+i*5]=W0->v;

  // off_diagonal elements
  M[0+2*5]=W0->d;

  M[4+2*5]=W0->d*W0->a*W0->a;

  M[2+4*5]=1./W0->d;


  // ===== Second Product

  d[0]=Dy->d;
  d[1]=Dy->u;
  d[2]=Dy->v;
  d[3]=Dy->w;
  d[4]=Dy->p;

  for(j=0;j<5;j++){
    for(i=0;i<5;i++){
      W[i]+=M[i+j*5]*d[j];
      }
  }

#ifdef WRADHYD
  W[5]+=W0->v*Dx->dX+W0->dX*Dx->v;
#ifdef HELIUM
  W[6]+=W0->v*Dx->dXHE+W0->dXHE*Dx->v;
  W[7]+=W0->v*Dx->dXXHE+W0->dXXHE*Dx->v;
#endif // HELIUM
#endif // WRADHYD

  // =====  building the C matrix

  memset(M,0,25*sizeof(REAL));

  // diagonal elements
  for(i=0;i<5;i++) M[i+i*5]=W0->w;

  // off_diagonal elements
  M[0+3*5]=W0->d;

  M[4+3*5]=W0->d*W0->a*W0->a;

  M[3+4*5]=1./W0->d;

  d[0]=Dz->d;
  d[1]=Dz->u;
  d[2]=Dz->v;
  d[3]=Dz->w;
  d[4]=Dz->p;

  // ===== Third Product

  for(j=0;j<5;j++){
    for(i=0;i<5;i++){
      W[i]+=M[i+j*5]*d[j];
      }
  }

#ifdef WRADHYD
  W[5]+=W0->w*Dx->dX+W0->w*Dx->dX;
#ifdef HELIUM
  W[6]+=W0->w*Dx->dXHE+W0->dXHE*Dx->w;
  W[7]+=W0->w*Dx->dXXHE+W0->dXXHE*Dx->w;
#endif // HELIUM
#endif // WRADHYD

  // ==== Final correction
  for(i=0;i<6;i++){
    W[i]*=(-dt/dx*0.5);
  }

  Wt->d=W[0];
  Wt->u=W[1];
  Wt->v=W[2];
  Wt->w=W[3];
  Wt->p=W[4];

#ifdef WRADHYD
  Wt->dX=W[5];
#ifdef HELIUM
  Wt->dXHE=W[6];
  Wt->dXXHE=W[7];
#endif // HELIUM
#endif // WRADHYD

}

//========================================================================================================================================
__device__ void  olddmatrix_jacobian(struct Wtype *W0, REAL dt,REAL dx,struct Wtype *Dx,struct Wtype *Dy,struct Wtype *Dz, struct Wtype *Wt){


  REAL M[25];
#ifdef HELIUM
  REAL W[8]={0.,0.,0.,0.,0.,0.,0.,0.};
#else
  REAL W[6]={0.,0.,0.,0.,0.,0.};
#endif // HELIUM
  REAL d[5];
  int i,j;
#ifdef WRADHYD
  REAL X;
#endif

  // =====  building the A matrix

  memset(M,0,25*sizeof(REAL));

  // diagonal elements
  for(i=0;i<5;i++) M[i+i*5]=W0->u;

  // off_diagonal elements
  M[0+1*5]=W0->d;

  M[4+1*5]=W0->d*W0->a*W0->a;

  M[1+4*5]=1./W0->d;


  // ===== First Product

  d[0]=Dx->d;
  d[1]=Dx->u;
  d[2]=Dx->v;
  d[3]=Dx->w;
  d[4]=Dx->p;

  for(j=0;j<5;j++){
    for(i=0;i<5;i++){
      W[i]+=M[i+j*5]*d[j];
    }
  }

#ifdef WRADHYD
  W[5]+=W0->u*Dx->dX+W0->dX*Dx->u;
#ifdef HELIUM
  W[6]+=W0->u*Dx->dXHE+W0->dXHE*Dx->u;
  W[7]+=W0->u*Dx->dXXHE+W0->dXXHE*Dx->u;
#endif // HELIUM
#endif // WRADHYD

  // =====  building the B matrix

  memset(M,0,25*sizeof(REAL));

  // diagonal elements
  for(i=0;i<5;i++) M[i+i*5]=W0->v;

  // off_diagonal elements
  M[0+2*5]=W0->d;

  M[4+2*5]=W0->d*W0->a*W0->a;

  M[2+4*5]=1./W0->d;


  // ===== Second Product

  d[0]=Dy->d;
  d[1]=Dy->u;
  d[2]=Dy->v;
  d[3]=Dy->w;
  d[4]=Dy->p;

  for(j=0;j<5;j++){
    for(i=0;i<5;i++){
      W[i]+=M[i+j*5]*d[j];
      }
  }

#ifdef WRADHYD
  W[5]+=W0->v*Dx->dX+W0->dX*Dx->v;
#ifdef HELIUM
  W[6]+=W0->v*Dx->dXHE+W0->dXHE*Dx->v;
  W[7]+=W0->v*Dx->dXXHE+W0->dXXHE*Dx->v;
#endif // HELIUM
#endif // WRADHYD

  // =====  building the C matrix

  memset(M,0,25*sizeof(REAL));

  // diagonal elements
  for(i=0;i<5;i++) M[i+i*5]=W0->w;

  // off_diagonal elements
  M[0+3*5]=W0->d;

  M[4+3*5]=W0->d*W0->a*W0->a;

  M[3+4*5]=1./W0->d;

  d[0]=Dz->d;
  d[1]=Dz->u;
  d[2]=Dz->v;
  d[3]=Dz->w;
  d[4]=Dz->p;

  // ===== Third Product

  for(j=0;j<5;j++){
    for(i=0;i<5;i++){
      W[i]+=M[i+j*5]*d[j];
      }
  }

#ifdef WRADHYD
  W[5]+=W0->w*Dx->dX+W0->w*Dx->dX;
#ifdef HELIUM
  W[6]+=W0->w*Dx->dXHE+W0->dXHE*Dx->w;
  W[7]+=W0->w*Dx->dXXHE+W0->dXXHE*Dx->w;
#endif // HELIUM
#endif // WRADHYD

  // ==== Final correction
  for(i=0;i<6;i++){
    W[i]*=(-dt/dx*0.5);
  }

  Wt->d=W[0];
  Wt->u=W[1];
  Wt->v=W[2];
  Wt->w=W[3];
  Wt->p=W[4];

#ifdef WRADHYD
  Wt->dX=W[5];
#ifdef HELIUM
  Wt->dXHE=W[6];
  Wt->dXXHE=W[7];
#endif // HELIUM
#endif // WRADHYD

}

__device__ void dMUSCL_BOUND2(struct HGRID *stencil, int ioct, int icell, struct Wtype *Wi,REAL dt,REAL dx){

	  struct OCT * oct;

	  struct Wtype *W0;
	  struct Wtype *Wp;
	  struct Wtype *Wm;
	  struct Wtype Dp,Dm;
	  struct Wtype D[3];
	  struct Wtype Wt;
	  int inei2;
	  int vcell[6],vnei[6];
	  int dir;
	  int idir;
	  int shift;

#ifdef WGRAV
	  REAL f[3];
	  struct Utype S;
	  struct Utype U;
#endif // WGRAV

	  getcellnei_gpu_hydro(icell, vnei, vcell); // we get the neighbors

	  W0=&(stencil->oct[ioct].cell[icell].field);

	  // Limited Slopes
	  shift=1;
	  for(dir=0;dir<3;dir++){

	    inei2=2*dir;
	    if(vnei[inei2]==6){
	      Wm=&(stencil->oct[ioct].cell[vcell[inei2]].field);
	    }
	    else{
	      Wm=&(stencil->oct[ioct-shift].cell[vcell[inei2]].field);
	    }

	    inei2=2*dir+1;
	    if(vnei[inei2]==6){
	      Wp=&(stencil->oct[ioct].cell[vcell[inei2]].field);
	    }
	    else{
	      Wp=&(stencil->oct[ioct+shift].cell[vcell[inei2]].field);
	    }

	    ddiffW(Wp,W0,&Dp);
	    ddiffW(W0,Wm,&Dm);

	    dminmod_W(&Dm,&Dp,D+dir);
	    shift*=3;
	  }


	  // build jacobian matrix product

	  dmatrix_jacobian(W0,dt,dx,&D[0],&D[1],&D[2],&Wt); // Here Wt contains the evolution of the state

	  // READY TO EVOLVE EXTRAPOLATED VALUE

	  REAL ix[]={-0.5,0.5,0.0,0.0,0.0,0.0};
	  REAL iy[]={0.0,0.0,-0.5,0.5,0.0,0.0};
	  REAL iz[]={0.0,0.0,0.0,0.0,-0.5,0.5};

#ifdef WGRAV
#ifndef NOCOUPLE
	  memcpy(f,stencil->oct[ioct].cell[icell].f,sizeof(REAL)*3);

#ifdef CONSERVATIVE
	  S.d =0.;
	  S.du=-W0->d*f[0]*0.5*dt;
	  S.dv=-W0->d*f[1]*0.5*dt;
	  S.dw=-W0->d*f[2]*0.5*dt;
	  S.E =-(W0->d*W0->u*f[0]+W0->d*W0->v*f[1]+W0->d*W0->w*f[2])*dt*0.5;
#endif // CONSERVATIVE

#endif // NOCOUPLE
#endif // WGRAV
	  for(idir=0;idir<6;idir++){
	    Wi[idir].d = W0->d+ix[idir]*D[0].d+iy[idir]*D[1].d+iz[idir]*D[2].d+Wt.d;
	    Wi[idir].u = W0->u+ix[idir]*D[0].u+iy[idir]*D[1].u+iz[idir]*D[2].u+Wt.u;
	    Wi[idir].v = W0->v+ix[idir]*D[0].v+iy[idir]*D[1].v+iz[idir]*D[2].v+Wt.v;
	    Wi[idir].w = W0->w+ix[idir]*D[0].w+iy[idir]*D[1].w+iz[idir]*D[2].w+Wt.w;
	    Wi[idir].p = FMAX(W0->p+ix[idir]*D[0].p+iy[idir]*D[1].p+iz[idir]*D[2].p+Wt.p,PMIN);
#ifdef WRADHYD
	    Wi[idir].dX = W0->dX+ix[idir]*D[0].dX+iy[idir]*D[1].dX+iz[idir]*D[2].dX+Wt.dX;
#ifdef HELIUM
	    Wi[idir].dXHE = W0->dXHE+ix[idir]*D[0].dXHE+iy[idir]*D[1].dXHE+iz[idir]*D[2].dXHE+Wt.dXHE;
	    Wi[idir].dXXHE = W0->dXXHE+ix[idir]*D[0].dXXHE+iy[idir]*D[1].dXXHE+iz[idir]*D[2].dXXHE+Wt.dXXHE;
#endif // HELIUM
#endif // WRADHYD
/* 	    if(Wi[idir].d<0) { */
/* 	      printf("neg d in extrapolation %e %e %e %e %e\n",Wi[idir].d,W0->d,D[0].d,D[1].d,D[2].d); */
/* 	      abort(); */
/* 	      } */
	    //if(Wi[idir].p==PMIN) printf("%e %e \n",W0->p,W0->p+ix[idir]*D[0].p+iy[idir]*D[1].p+iz[idir]*D[2].p+Wt.p);


#ifdef WGRAV
#ifndef NOCOUPLE

#ifdef PRIMITIVE
	    Wi[idir].u+=-f[0]*0.5*dt;
	    Wi[idir].v+=-f[1]*0.5*dt;
	    Wi[idir].w+=-f[2]*0.5*dt;
#endif // PRIMITIVE

#ifdef CONSERVATIVE
 	    W2U(&Wi[idir],&U);
	    U.d  +=S.d;
	    U.du +=S.du;
	    U.dv +=S.dv;
	    U.dw +=S.dw;
	    U.E  +=S.E;
	    U2W(&U,&Wi[idir]);
#endif // CONSERVATIVE

#endif // NOCOUPLE
#endif // WGRAV

	    dgetE(Wi+idir);
	    Wi[idir].a=SQRT(GAMMA*Wi[idir].p/Wi[idir].d);

#ifdef WRADHYD
	    REAL X0=W0->dX/(W0->d*(1.-YHE));
 	    Wi[idir].dX=Wi[idir].d*(1.-YHE)*X0;
#ifdef HELIUM
	    REAL XHE0=W0->dXHE/(W0->d*(YHE));
 	    Wi[idir].dXHE=Wi[idir].d*(YHE)*XHE0;
	    REAL XXHE0=W0->dXXHE/(W0->d*(YHE));
 	    Wi[idir].dXXHE=Wi[idir].d*(YHE)*XXHE0;
#endif // HELIUM
#endif // WRADHYD
	  }




}

// ==============================================
__device__ void olddMUSCL_BOUND2(struct HGRID *stencil, int ioct, int icell, struct Wtype *Wi,REAL dt,REAL dx){ 
  
  struct Wtype *W0;
  struct Wtype *Wp;
  struct Wtype *Wm;
  struct Wtype Dp,Dm;
  struct Wtype D[3];
  struct Wtype Wt;
  int inei2;
  int vcell[6],vnei[6];
  int dir;
  int idir;
  int shift;

#ifdef WGRAV
	  REAL f[3];
#ifdef CONSERVATIVE
	  struct Utype S;
	  struct Utype U;
#endif
#endif

	  getcellnei_gpu_hydro(icell, vnei, vcell); // we get the neighbors
	  
	  W0=&(stencil->oct[ioct].cell[icell].field);
	
	  // Limited Slopes
	  shift=1;
	  for(dir=0;dir<3;dir++){
	    
	    inei2=2*dir;
	    if(vnei[inei2]==6){
	      Wm=&(stencil->oct[ioct].cell[vcell[inei2]].field);
	    }
	    else{
	      Wm=&(stencil->oct[ioct-shift].cell[vcell[inei2]].field);
	    }

	    inei2=2*dir+1;
	    if(vnei[inei2]==6){
	      Wp=&(stencil->oct[ioct].cell[vcell[inei2]].field);
	    }
	    else{
	      Wp=&(stencil->oct[ioct+shift].cell[vcell[inei2]].field);
	    }

	    ddiffW(Wp,W0,&Dp); 
	    ddiffW(W0,Wm,&Dm); 
	    
	    dminmod_W(&Dm,&Dp,D+dir);
	    shift*=3;
	  }


	  // build jacobian matrix product
	  
	  dmatrix_jacobian(W0,dt,dx,&D[0],&D[1],&D[2],&Wt); // Here Wt contains the evolution of the state

	  
	  // READY TO EVOLVE EXTRAPOLATED VALUE

	  REAL ix[]={-0.5,0.5,0.0,0.0,0.0,0.0};
	  REAL iy[]={0.0,0.0,-0.5,0.5,0.0,0.0};
	  REAL iz[]={0.0,0.0,0.0,0.0,-0.5,0.5};

#ifdef WGRAV
#ifndef NOCOUPLE
	  memcpy(f,stencil->oct[ioct].cell[icell].f,sizeof(REAL)*3);
#ifdef CONSERVATIVE
	  S.d =0.;
	  S.du=-W0->d*f[0]*0.5*dt;
	  S.dv=-W0->d*f[1]*0.5*dt;
	  S.dw=-W0->d*f[2]*0.5*dt;
	  S.E =-(W0->d*W0->u*f[0]+W0->d*W0->v*f[1]+W0->d*W0->w*f[2])*dt*0.5;
#endif

#endif
#endif
	  for(idir=0;idir<6;idir++){
	    Wi[idir].d = W0->d+ix[idir]*D[0].d+iy[idir]*D[1].d+iz[idir]*D[2].d+Wt.d;
	    Wi[idir].u = W0->u+ix[idir]*D[0].u+iy[idir]*D[1].u+iz[idir]*D[2].u+Wt.u;
	    Wi[idir].v = W0->v+ix[idir]*D[0].v+iy[idir]*D[1].v+iz[idir]*D[2].v+Wt.v;
	    Wi[idir].w = W0->w+ix[idir]*D[0].w+iy[idir]*D[1].w+iz[idir]*D[2].w+Wt.w;
	    Wi[idir].p = FMAX(W0->p+ix[idir]*D[0].p+iy[idir]*D[1].p+iz[idir]*D[2].p+Wt.p,PMIN);
#ifdef WRADHYD
	    Wi[idir].dX = W0->dX+ix[idir]*D[0].dX+iy[idir]*D[1].dX+iz[idir]*D[2].dX+Wt.dX;
#ifdef HELIUM
	    Wi[idir].dXHE = W0->dXHE+ix[idir]*D[0].dXHE+iy[idir]*D[1].dXHE+iz[idir]*D[2].dXHE+Wt.dXHE;
	    Wi[idir].dXXHE = W0->dXXHE+ix[idir]*D[0].dXXHE+iy[idir]*D[1].dXXHE+iz[idir]*D[2].dXXHE+Wt.dXXHE;
#endif
#endif

	     /* if(Wi[idir].p<0) abort(); */
	    /* if(Wi[idir].d<0) abort(); */


#ifdef WGRAV
#ifndef NOCOUPLE

#ifdef PRIMITIVE
	    Wi[idir].u+=-f[0]*0.5*dt;
	    Wi[idir].v+=-f[1]*0.5*dt;
	    Wi[idir].w+=-f[2]*0.5*dt;
#endif

#ifdef CONSERVATIVE
 	    dW2U(&Wi[idir],&U);
	    U.d  +=S.d;
	    U.du +=S.du;
	    U.dv +=S.dv;
	    U.dw +=S.dw;
	    U.E  +=S.E;
	    dU2W(&U,&Wi[idir]);
#endif

#endif
#endif
	    dgetE(Wi+idir);
	    Wi[idir].a=SQRT(GAMMA*Wi[idir].p/Wi[idir].d);
#ifdef WRADHYD
	    REAL X0=W0->dX/(W0->d*(1.-YHE));
 	    Wi[idir].dX=Wi[idir].d*(1.-YHE)*X0; 
#ifdef HELIUM
	    REAL XHE0=W0->dXHE/(W0->d*(YHE));
 	    Wi[idir].dXHE=Wi[idir].d*(YHE)*XHE0; 
	    REAL XXHE0=W0->dXXHE/(W0->d*(YHE));
 	    Wi[idir].dXXHE=Wi[idir].d*(YHE)*XXHE0; 
#endif
#endif 
	  }



	  
}

//========================================================================================
__device__ REAL dfrootprime(REAL p, struct Wtype1D *WL, struct Wtype1D *WR)
{
  
  REAL fL,fR;
  REAL AL,AR,BL,BR;

  AL=2./((GAMMA+1.)*WL->d);
  AR=2./((GAMMA+1.)*WR->d);
  
  BL=(GAMMA-1.)/(GAMMA+1.)*WL->p;
  BR=(GAMMA-1.)/(GAMMA+1.)*WR->p;

  fL=(p>WL->p?SQRT(AL/(BL+p))*(1.-(p-WL->p)/(2.*(BL+p))):POW(p/WL->p,-(GAMMA+1)/(2.*GAMMA))/(WL->d*WL->a));
  fR=(p>WR->p?SQRT(AR/(BR+p))*(1.-(p-WR->p)/(2.*(BR+p))):POW(p/WR->p,-(GAMMA+1)/(2.*GAMMA))/(WR->d*WR->a));

  return fL+fR;
}


// ------------------------------------

__device__ REAL dfroot(REAL p, struct Wtype1D *WL, struct Wtype1D *WR, REAL *u)
{
  
  REAL fL,fR;
  REAL AL,AR,BL,BR;
  REAL Deltau;

  AL=2./((GAMMA+1.)*WL->d);
  AR=2./((GAMMA+1.)*WR->d);
  
  BL=(GAMMA-1.)/(GAMMA+1.)*WL->p;
  BR=(GAMMA-1.)/(GAMMA+1.)*WR->p;

  fL=(p>WL->p?(p-WL->p)*SQRT(AL/(BL+p)):2.*WL->a/(GAMMA-1.)*(POW(p/WL->p,(GAMMA-1)/(2.*GAMMA))-1.));
  fR=(p>WR->p?(p-WR->p)*SQRT(AR/(BR+p)):2.*WR->a/(GAMMA-1.)*(POW(p/WR->p,(GAMMA-1)/(2.*GAMMA))-1.));
  
  Deltau=WR->u-WL->u;
  *u=0.5*(WL->u+WR->u)+0.5*(fR-fL);

  return fL+fR+Deltau;
}


//========================================================================================
//========================================================================================
__device__ REAL dfindPressure(struct Wtype1D *WL, struct Wtype1D *WR, int *niter, REAL *u)
{

  REAL ptr,pts,ppv;
  REAL ptr0,pts0,ppv0;
  REAL p,porg,dp;
  int i;
  REAL err;
  REAL unsurz=(2.0*GAMMA)/(GAMMA-1.0);
  REAL AL,AR,BL,BR,GL,GR;
  REAL pmin,pmax;
  REAL u2;

  pmin=FMIN(WL->p,WR->p);
  pmax=FMAX(WL->p,WR->p);
  
  // EXACT SOLVER

  // hybrid guess for pressure

  AL=2./((GAMMA+1.)*WL->d);
  AR=2./((GAMMA+1.)*WR->d);
  
  BL=(GAMMA-1.)/(GAMMA+1.)*WL->p;
  BR=(GAMMA-1.)/(GAMMA+1.)*WR->p;

  ppv0=0.5*(WL->p+WR->p)-0.125*(WR->u-WL->u)*(WR->d+WL->d)*(WR->a+WL->a);
  ptr0=POW((WL->a+WR->a-0.5*(GAMMA-1)*(WR->u-WL->u))/(WL->a/POW(WL->p,1./unsurz)+WR->a/POW(WR->p,1./unsurz)),unsurz);

  ppv=FMAX(ERRTOL,ppv0);
  ptr=FMAX(ERRTOL,ptr0);
  
  GL=SQRT(AL/(ppv+BL));
  GR=SQRT(AR/(ppv+BR));

  pts0=(GL*WL->p+GR*WR->p-(WR->u-WL->u))/(GL+GR);
  pts=FMAX(ERRTOL,pts0);


  if(((pmax/pmin)<2.0)&&((pmin<=ppv)&&(ppv<=pmax))){
      p=ppv;
    }
  else{
    if(ppv<pmin){
      p=ptr;
    }
    else{
      p=pts;
    }
  }


  //p=0.5*(WL->p+WR->p);
  //p=FMAX(p,ERRTOL);

  *niter=0;
  for(i=0;i<NITERMAX;i++)
    {
      dp=dfroot(p,WL,WR,&u2)/dfrootprime(p,WL,WR);

      if(FABS(dp)<ERRTOL) break;
      while((p-dp)<0){ 
       	dp=dp*0.5; 
      } 

      porg=p;
      p=p-dp;
      err=2.*FABS(p-porg)/(FABS(p+porg));
      *niter=*niter+1;
      if(err<ERRTOL) break;
      if(dfroot(p,WL,WR,&u2)<ERRTOL) break;
    }

  dfroot(p,WL,WR,&u2); // last calculation to get u;

  *u=(REAL)u2;
  return p;
}


//========================================================================================
//========================================================================================
__device__ REAL dfindPressure_Hybrid(struct Wtype1D *WL, struct Wtype1D *WR, int *niter, REAL *ustar){
  REAL ppvrs;
  REAL dbar,abar;
  REAL pmax,pmin,pstar;
  REAL AL,AR,BL,BR,GL,GR;
  dbar=0.5*(WL->d+WR->d);
  abar=0.5*(WL->a+WR->a);
  ppvrs=0.5*((WL->p+WR->p)+(WL->u-WR->u)*dbar*abar);
  pmax=FMAX(WL->p,WR->p);
  pmin=FMIN(WL->p,WR->p);
  pstar=ppvrs;
  
  //printf("dbar=%e abar=%e ppvrs=%e pmax=%e pmin=%e pstar=%e\n",dbar,abar,ppvrs,pmax,pmin,pstar);

  if(((pmax/pmin)<2.)&&((pmin<pstar)&&(pstar<pmax))){
    // PVRS CASE
    pstar=ppvrs;
    *ustar=0.5*((WL->u+WR->u)+(WL->p-WR->p)/(dbar*abar));
  }
  else{
    if(pstar<pmin){
      //TRRS CASE
      REAL z=(GAMMA-1.)/(2.*GAMMA);
      REAL iz=(2.*GAMMA)/(GAMMA-1.);
      pstar=POW((WL->a+WR->a-(GAMMA-1.)/2.*(WR->u-WL->u))/(WL->a/POW(WL->p,z)+WR->a/POW(WR->p,z)),iz);
      *ustar=WL->u-2.*WL->a/(GAMMA-1.)*(POW(pstar/WL->p,z)-1.);
    }
    else{
      //TSRS CASE
      REAL p0;
      p0=FMAX(0.,ppvrs);
      
      AL=2./((GAMMA+1.)*WL->d);
      AR=2./((GAMMA+1.)*WR->d);
      
      BL=(GAMMA-1.)/(GAMMA+1.)*WL->p;
      BR=(GAMMA-1.)/(GAMMA+1.)*WR->p;

      GL=SQRT(AL/(p0+BL));
      GR=SQRT(AR/(p0+BR));

      pstar=(GL*WL->p+GR*WR->p-(WR->u-WL->u))/(GL+GR);
      *ustar=0.5*((WL->u+WR->u)+(pstar-WR->p)*GR-(pstar-WL->p)*GL);
    }
  }

  return pstar;

}




//====================================================================
__device__ void dspeedestimateX_HLLC(struct Wtype *WL,struct Wtype *WR, REAL *SL, REAL *SR, REAL *pstar, REAL *ustar){

  REAL qL,qR;
  struct Wtype1D WLloc;
  struct Wtype1D WRloc;
  int n;

  WLloc.d=WL->d;
  WLloc.u=WL->u;
  WLloc.p=WL->p;
  WLloc.a=SQRT(GAMMA*WLloc.p/WLloc.d);
  
  WRloc.d=WR->d;
  WRloc.u=WR->u;
  WRloc.p=WR->p;
  WRloc.a=SQRT(GAMMA*WRloc.p/WRloc.d);

  //printf("%e %e %e %e ||| %e %e %e %e\n",WLloc.d,WLloc.u,WLloc.p,WLloc.a,WLloc.d,WRloc.u,WRloc.p,WRloc.a);

#if 1
  (*pstar)= dfindPressure_Hybrid(&WLloc,&WRloc,&n,ustar);
  if((*pstar)<0) (*pstar)=dfindPressure(&WLloc,&WRloc,&n,ustar);
  //if((*pstar)<=0) printf("shhh pstar=%e %e %d\n",*pstar,*ustar,n);

  qL=(*pstar<=WL->p?1.:SQRT(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WL->p-1.)));
  qR=(*pstar<=WR->p?1.:SQRT(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WR->p-1.)));
  
  *SL=WLloc.u-WLloc.a*qL;
  *SR=WRloc.u+WRloc.a*qR;
  if((*SL)>(*SR)){
    (*SL)=FMIN(WLloc.u-WLloc.a,WRloc.u-WRloc.a);
    (*SR)=FMAX(WLloc.u+WLloc.a,WRloc.u+WRloc.a);
  }
#endif
  
  //if(isnan(*ustar)) printf("Hehey\n");
}

//====================================================================

void __device__ dspeedestimateY_HLLC(struct Wtype *WL,struct Wtype *WR, REAL *SL, REAL *SR, REAL *pstar, REAL *ustar){

  REAL qL,qR;
  struct Wtype1D WLloc;
  struct Wtype1D WRloc;
  int n;

  WLloc.d=WL->d;
  WLloc.u=WL->v;
  WLloc.p=WL->p;
  WLloc.a=SQRT(GAMMA*WLloc.p/WLloc.d);
  
  WRloc.d=WR->d;
  WRloc.u=WR->v;
  WRloc.p=WR->p;
  WRloc.a=SQRT(GAMMA*WRloc.p/WRloc.d);

  (*pstar)=dfindPressure_Hybrid(&WLloc,&WRloc,&n,ustar);
  if((*pstar)<0) (*pstar)=dfindPressure(&WLloc,&WRloc,&n,ustar);
  //  if((*pstar)<0) abort();

  qL=(*pstar<=WL->p?1.:SQRT(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WL->p-1.)));
  qR=(*pstar<=WR->p?1.:SQRT(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WR->p-1.)));
  
  *SL=WLloc.u-WLloc.a*qL;
  *SR=WRloc.u+WRloc.a*qR;

  if((*SL)>(*SR)){
    (*SL)=FMIN(WLloc.u-WLloc.a,WRloc.u-WRloc.a);
    (*SR)=FMAX(WLloc.u+WLloc.a,WRloc.u+WRloc.a);
    //abort();
  }
  //  if((*SL)>(*SR)) abort();
  //if(isnan(*ustar)) printf("Hehey y\n");

}



//====================================================================

void __device__ dspeedestimateZ_HLLC(struct Wtype *WL,struct Wtype *WR, REAL *SL, REAL *SR, REAL *pstar, REAL *ustar){

  REAL qL,qR;
  struct Wtype1D WLloc;
  struct Wtype1D WRloc;
  int n;

  WLloc.d=WL->d;
  WLloc.u=WL->w;
  WLloc.p=WL->p;
  WLloc.a=SQRT(GAMMA*WLloc.p/WLloc.d);
  
  WRloc.d=WR->d;
  WRloc.u=WR->w;
  WRloc.p=WR->p;
  WRloc.a=SQRT(GAMMA*WRloc.p/WRloc.d);

  (*pstar)=dfindPressure_Hybrid(&WLloc,&WRloc,&n,ustar);
  if((*pstar)<0) (*pstar)=dfindPressure(&WLloc,&WRloc,&n,ustar);
  //if((*pstar)<0) abort();

  qL=(*pstar<=WL->p?1.:SQRT(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WL->p-1.)));
  qR=(*pstar<=WR->p?1.:SQRT(1.+(GAMMA+1.)/(2.*GAMMA)*((*pstar)/WR->p-1.)));
  
  *SL=WLloc.u-WLloc.a*qL;
  *SR=WRloc.u+WRloc.a*qR;
  if((*SL)>(*SR)){
    (*SL)=FMIN(WLloc.u-WLloc.a,WRloc.u-WRloc.a);
    (*SR)=FMAX(WLloc.u+WLloc.a,WRloc.u+WRloc.a);
    //abort();
  }
  //if((*SL)>(*SR)) abort();
  //if(isnan(*ustar)) printf("Hehey z\n");

}


// =============================================================================================

__global__  void dhydroM_sweepZ(struct HGRID *stencil,int nread,REAL dx, REAL dt){

  int inei,icell,iface;
  int i;
  int vnei[6],vcell[6];

  REAL FL[NVAR],FR[NVAR];
  struct Utype Uold;
  struct Wtype Wold;
  REAL pstar,ustar;

  struct Wtype WT[6]; // FOR MUSCL RECONSTRUCTION
  struct Wtype WC[6]; // FOR MUSCL RECONSTRUCTION

  struct Utype UC[2];
  struct Utype UN[2];
  struct Wtype WN[2];

  int ioct[7]={12,14,10,16,4,22,13};
  int idxnei[6]={1,0,3,2,5,4};

  struct Wtype *curcell;

  REAL SL,SR;

  int ffact[2]={0,0};
  REAL fact;

  struct Utype Us; 
  dinitUtype(&Us);
  REAL ebar;
  REAL ecen=0.;
  REAL divu,divuloc;

  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;

  i=bx*blockDim.x+tx;

  for(icell=0;icell<8;icell++){ // we scan the cells
     getcellnei_gpu_hydro(icell, vnei, vcell); // we get the neighbors

    if(i<nread){

      memset(FL,0,sizeof(REAL)*NVAR);
      memset(FR,0,sizeof(REAL)*NVAR);

#if 1
      // Getting the original state ===========================

      curcell=&(stencil[i].oct[ioct[6]].cell[icell].field);

      divu=stencil[i].New.cell[icell].divu;

      Wold.d=curcell->d;
      Wold.u=curcell->u;
      Wold.v=curcell->v;
      Wold.w=curcell->w;
      Wold.p=curcell->p;
      Wold.a=SQRT(GAMMA*Wold.p/Wold.d);


      dW2U(&Wold,&Uold); // primitive -> conservative


      REAL eold=Uold.eint;
      /* // MUSCL STATE RECONSTRUCTION */
      memset(ffact,0,sizeof(int)*2);

      dMUSCL_BOUND2(stencil+i, 13, icell, WC,dt,dx);// central


      for(iface=0;iface<2;iface++){
	inei=iface+4;
	memcpy(WC+iface,WC+inei,sizeof(struct Wtype)); // moving the data towards idx=0,1
	//memcpy(WC+iface,&(stencil[i].oct[13].cell[inei].field),sizeof(struct Wtype)); // moving the data towards idx=0,1 //HACK

	dW2U(WC+iface,UC+iface);
      }

      // Neighbor MUSCL reconstruction
      for(iface=0;iface<2;iface++){
	inei=iface+4;
	dMUSCL_BOUND2(stencil+i, ioct[vnei[inei]], vcell[inei], WT,dt,dx);//


	memcpy(WN+iface,WT+idxnei[inei],sizeof(struct Wtype));
	//memcpy(WN+iface,&(stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].field),sizeof(struct Wtype)); //HACK
	dW2U(WN+iface,UN+iface);

	if(!stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].split){
	  ffact[iface]=1; // we cancel the contriubtion of split neighbors
	}

      }


      // Z DIRECTION =========================================================================

      // --------- solving the Riemann Problems BOTTOM

      // Switching to Split description

      /* 	// =========================================== */

#ifdef RIEMANN_HLLC
      dspeedestimateZ_HLLC(&WN[0],&WC[0],&SL,&SR,&pstar,&ustar);

      if(SL>=0.){
	dgetflux_Z(&UN[0],FL);
	memcpy(&Us,&UN[0],sizeof(struct Utype));

      }
      else if(SR<=0.){
	dgetflux_Z(&UC[0],FL);
	memcpy(&Us,&UC[0],sizeof(struct Utype));
      }
      else if((SL<0.)&&(ustar>=0.)){
	dgetflux_Z(&UN[0],FL);
	fact=WN[0].d*(SL-WN[0].w)/(SL-ustar);
	FL[0]+=(fact*1.                                                                      -UN[0].d )*SL;
	FL[1]+=(fact*WN[0].u                                                                 -UN[0].du)*SL;
	FL[2]+=(fact*WN[0].v                                                                 -UN[0].dv)*SL;
	FL[3]+=(fact*ustar                                                                   -UN[0].dw)*SL;
	FL[4]+=(fact*(UN[0].E/UN[0].d+(ustar-WN[0].w)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].w))))-UN[0].E )*SL;

	Us.d =(fact*1.);
	Us.du=(fact*WN[0].u);
	Us.dv=(fact*WN[0].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UN[0].E/UN[0].d+(ustar-WN[0].w)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].w)))));

#ifdef WRADHYD
	FL[6]+=(fact*WN[0].dX/WN[0].d                                                                 -UN[0].dX)*SL;
 #ifdef HELIUM
	FL[7]+=(fact*WN[0].dXHE/WN[0].d                                                                 -UN[0].dXHE)*SL;
	FL[8]+=(fact*WN[0].dXXHE/WN[0].d                                                                 -UN[0].dXXHE)*SL;
#endif // HELIUM
#endif // WRADHYD
      }
      else if((ustar<=0.)&&(SR>0.)){
	dgetflux_Z(&UC[0],FL);
	fact=WC[0].d*(SR-WC[0].w)/(SR-ustar);
	FL[0]+=(fact*1.                                                                      -UC[0].d )*SR;
	FL[1]+=(fact*WC[0].u                                                                 -UC[0].du)*SR;
	FL[2]+=(fact*WC[0].v                                                                 -UC[0].dv)*SR;
	FL[3]+=(fact*ustar                                                                   -UC[0].dw)*SR;
	FL[4]+=(fact*(UC[0].E/UC[0].d+(ustar-WC[0].w)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].w))))-UC[0].E )*SR;

	Us.d =(fact*1.);
	Us.du=(fact*WC[0].u);
	Us.dv=(fact*WC[0].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UC[0].E/UC[0].d+(ustar-WC[0].w)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].w)))));

#ifdef WRADHYD
	FL[6]+=(fact*WC[0].dX/WC[0].d                                                                 -UC[0].dX)*SR;
#ifdef HELIUM
	FL[7]+=(fact*WC[0].dXHE/WC[0].d                                                                 -UC[0].dXHE)*SR;
	FL[8]+=(fact*WC[0].dXXHE/WC[0].d                                                                 -UC[0].dXXHE)*SR;
#endif // HELIUM
#endif // WRADHYD
      }

      ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d);
      divuloc=(GAMMA-1.)*(Us.dw/Us.d)*eold;
      FL[5]=(Us.dw/Us.d*ebar);
      divu+=-divuloc;

#endif // RIEMANN_HLLC
      // ===========================================



      // --------- solving the Riemann Problems TOP


      // Switching to Split description

      //=====================================================

#ifdef RIEMANN_HLLC
      dspeedestimateZ_HLLC(&WC[1],&WN[1],&SL,&SR,&pstar,&ustar);

      if(SL>=0.){
	dgetflux_Z(&UC[1],FR);
	memcpy(&Us,&UC[1],sizeof(struct Utype));

      }
      else if(SR<=0.){
	dgetflux_Z(&UN[1],FR);
	memcpy(&Us,&UN[1],sizeof(struct Utype));
      }
      else if((SL<0.)&&(ustar>=0.)){
	dgetflux_Z(&UC[1],FR);
	fact=WC[1].d*(SL-WC[1].w)/(SL-ustar);
	FR[0]+=(fact*1.                                                                      -UC[1].d )*SL;
	FR[1]+=(fact*WC[1].u                                                                 -UC[1].du)*SL;
	FR[2]+=(fact*WC[1].v                                                                 -UC[1].dv)*SL;
	FR[3]+=(fact*ustar                                                                   -UC[1].dw)*SL;
	FR[4]+=(fact*(UC[1].E/UC[1].d+(ustar-WC[1].w)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].w))))-UC[1].E )*SL;

	Us.d =(fact*1.);
	Us.du=(fact*WC[1].u);
	Us.dv=(fact*WC[1].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UC[1].E/UC[1].d+(ustar-WC[1].w)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].w)))));

#ifdef WRADHYD
	FR[6]+=(fact*WC[1].dX/WC[1].d                                                                 -UC[1].dX)*SL;
#ifdef HELIUM
	FR[7]+=(fact*WC[1].dXHE/WC[1].d                                                                 -UC[1].dXHE)*SL;
	FR[8]+=(fact*WC[1].dXXHE/WC[1].d                                                                 -UC[1].dXXHE)*SL;
#endif // HELIUM
#endif // WRADHYD
      }
      else if((ustar<=0.)&&(SR>0.)){
	dgetflux_Z(&UN[1],FR);
	fact=WN[1].d*(SR-WN[1].w)/(SR-ustar);
	FR[0]+=(fact*1.                                                                      -UN[1].d )*SR;
	FR[1]+=(fact*WN[1].u                                                                 -UN[1].du)*SR;
	FR[2]+=(fact*WN[1].v                                                                 -UN[1].dv)*SR;
	FR[3]+=(fact*ustar                                                                   -UN[1].dw)*SR;
	FR[4]+=(fact*(UN[1].E/UN[1].d+(ustar-WN[1].w)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].w))))-UN[1].E )*SR;

	Us.d =(fact*1.);
	Us.du=(fact*WN[1].u);
	Us.dv=(fact*WN[1].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UN[1].E/UN[1].d+(ustar-WN[1].w)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].w)))));

#ifdef WRADHYD
	FR[6]+=(fact*WN[1].dX/WN[1].d                                                                 -UN[1].dX)*SR;
#ifdef HELIUM
	FR[7]+=(fact*WN[1].dXHE/WN[1].d                                                                 -UN[1].dXHE)*SR;
	FR[8]+=(fact*WN[1].dXXHE/WN[1].d                                                                 -UN[1].dXXHE)*SR;
#endif // HELIUM
#endif // WRADHYD
      }

      ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d);
      divuloc=(GAMMA-1.)*(Us.dw/Us.d)*eold;
      FR[5]=(Us.dw/Us.d*ebar);
      divu+= divuloc;


#endif // RIEMANN_HLLC


      //========================= copy the fluxes

      // Cancelling the fluxes from splitted neighbours

#endif

      for(iface=0;iface<NVAR;iface++) FL[iface]*=ffact[0];
      for(iface=0;iface<NVAR;iface++) FR[iface]*=ffact[1];


      memcpy(stencil[i].New.cell[icell].flux+4*NVAR,FL,sizeof(REAL)*NVAR);
      memcpy(stencil[i].New.cell[icell].flux+5*NVAR,FR,sizeof(REAL)*NVAR);

      stencil[i].New.cell[icell].divu=divu;

      // ready for the next cell
    }
    //ready for the next oct
  }

}


__global__ void olddhydroM_sweepZ(struct HGRID *stencil, int nread,REAL dx, REAL dt){

  int inei,icell,iface;
  int i;
  int vnei[6],vcell[6];

  REAL FL[NVAR],FR[NVAR];
  struct Utype Uold;
  struct Wtype Wold;
  REAL pstar,ustar;

  struct Wtype WT[6]; // FOR MUSCL RECONSTRUCTION
  struct Wtype WC[6]; // FOR MUSCL RECONSTRUCTION

  struct Utype UC[2];
  struct Utype UN[2];
  struct Wtype WN[2];

  int ioct[7]={12,14,10,16,4,22,13};
  int idxnei[6]={1,0,3,2,5,4};

  struct Wtype *curcell;

  REAL SL,SR;
  
  int ffact[2]={0,0};
  REAL fact;

  struct Utype Us;
  REAL ebar;
  REAL divu,divuloc;

  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;

  i=bx*blockDim.x+tx;
  if(i<nread){
  for(icell=0;icell<8;icell++){ // we scan the cells
    getcellnei_gpu_hydro(icell, vnei, vcell); // we get the neighbors
      
      
      memset(FL,0,sizeof(REAL)*NVAR);
      memset(FR,0,sizeof(REAL)*NVAR);

#if 1
      // Getting the original state ===========================
      
      curcell=&(stencil[i].oct[ioct[6]].cell[icell].field);

#ifdef DUAL_E
      divu=stencil[i].New.cell[icell].divu;
#endif      

      Wold.d=curcell->d;
      Wold.u=curcell->u;
      Wold.v=curcell->v;
      Wold.w=curcell->w;
      Wold.p=curcell->p;
      Wold.a=SQRT(GAMMA*Wold.p/Wold.d);


      dW2U(&Wold,&Uold); // primitive -> conservative

      REAL eold=Uold.eint;

      /* // MUSCL STATE RECONSTRUCTION */
      memset(ffact,0,sizeof(int)*2);

      dMUSCL_BOUND2(stencil+i, 13, icell, WC,dt,dx);// central

      for(iface=0;iface<2;iface++){
	inei=iface+4;
	memcpy(WC+iface,WC+inei,sizeof(struct Wtype)); // moving the data towards idx=0,1
	//memcpy(WC+iface,&(stencil[i].oct[13].cell[inei].field),sizeof(struct Wtype)); // moving the data towards idx=0,1 //HACK
	dW2U(WC+iface,UC+iface);
      }

      // Neighbor MUSCL reconstruction
      for(iface=0;iface<2;iface++){
	inei=iface+4;
	dMUSCL_BOUND2(stencil+i, ioct[vnei[inei]], vcell[inei], WT,dt,dx);// 

	memcpy(WN+iface,WT+idxnei[inei],sizeof(struct Wtype)); 

	//memcpy(WN+iface,&(stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].field),sizeof(struct Wtype));  //HACK
	
	dW2U(WN+iface,UN+iface);
	
	if(!stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].split){
	  ffact[iface]=1; // we cancel the contriubtion of split neighbors
	}

      }


      // Z DIRECTION =========================================================================
      
      // --------- solving the Riemann Problems BOTTOM

      // Switching to Split description

      /* 	// =========================================== */

#ifdef RIEMANN_HLLC
      dspeedestimateZ_HLLC(&WN[0],&WC[0],&SL,&SR,&pstar,&ustar);

      if(SL>=0.){
	dgetflux_Z(&UN[0],FL);
	memcpy(&Us,&UN[0],sizeof(struct Utype));

      }
      else if(SR<=0.){
	dgetflux_Z(&UC[0],FL);
	memcpy(&Us,&UC[0],sizeof(struct Utype));
      }
      else if((SL<0.)&&(ustar>=0.)){
	dgetflux_Z(&UN[0],FL);
	fact=WN[0].d*(SL-WN[0].w)/(SL-ustar);
	FL[0]+=(fact*1.                                                                      -UN[0].d )*SL;
	FL[1]+=(fact*WN[0].u                                                                 -UN[0].du)*SL;
	FL[2]+=(fact*WN[0].v                                                                 -UN[0].dv)*SL;
	FL[3]+=(fact*ustar                                                                   -UN[0].dw)*SL;
	FL[4]+=(fact*(UN[0].E/UN[0].d+(ustar-WN[0].w)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].w))))-UN[0].E )*SL;

	Us.d =(fact*1.);
	Us.du=(fact*WN[0].u);
	Us.dv=(fact*WN[0].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UN[0].E/UN[0].d+(ustar-WN[0].w)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].w)))));

#ifdef WRADHYD
	FL[6]+=(fact*WN[0].dX/WN[0].d                                                                 -UN[0].dX)*SL;
 #ifdef HELIUM
	FL[7]+=(fact*WN[0].dXHE/WN[0].d                                                                 -UN[0].dXHE)*SL;
	FL[8]+=(fact*WN[0].dXXHE/WN[0].d                                                                 -UN[0].dXXHE)*SL;
#endif
#endif
	

      }
      else if((ustar<=0.)&&(SR>0.)){
	dgetflux_Z(&UC[0],FL);
	fact=WC[0].d*(SR-WC[0].w)/(SR-ustar);
	FL[0]+=(fact*1.                                                                      -UC[0].d )*SR;
	FL[1]+=(fact*WC[0].u                                                                 -UC[0].du)*SR;
	FL[2]+=(fact*WC[0].v                                                                 -UC[0].dv)*SR;
	FL[3]+=(fact*ustar                                                                   -UC[0].dw)*SR;
	FL[4]+=(fact*(UC[0].E/UC[0].d+(ustar-WC[0].w)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].w))))-UC[0].E )*SR;

	Us.d =(fact*1.);
	Us.du=(fact*WC[0].u);
	Us.dv=(fact*WC[0].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UC[0].E/UC[0].d+(ustar-WC[0].w)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].w)))));

#ifdef WRADHYD
	FL[6]+=(fact*WC[0].dX/WC[0].d                                                                 -UC[0].dX)*SR;
#ifdef HELIUM
	FL[7]+=(fact*WC[0].dXHE/WC[0].d                                                                 -UC[0].dXHE)*SR;
	FL[8]+=(fact*WC[0].dXXHE/WC[0].d                                                                 -UC[0].dXXHE)*SR;
#endif

#endif

      }

      ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
      divuloc=(GAMMA-1.)*(Us.dw/Us.d)*eold;
      FL[5]=(Us.dw/Us.d*ebar);
      divu+=-divuloc;

#endif
      // ===========================================



      // --------- solving the Riemann Problems TOP


      // Switching to Split description

      //=====================================================

#ifdef RIEMANN_HLLC
      dspeedestimateZ_HLLC(&WC[1],&WN[1],&SL,&SR,&pstar,&ustar);

      if(SL>=0.){
	dgetflux_Z(&UC[1],FR);
	memcpy(&Us,&UC[1],sizeof(struct Utype));

      }
      else if(SR<=0.){
	dgetflux_Z(&UN[1],FR);
	memcpy(&Us,&UN[1],sizeof(struct Utype));

      }
      else if((SL<0.)&&(ustar>=0.)){
	dgetflux_Z(&UC[1],FR);
	fact=WC[1].d*(SL-WC[1].w)/(SL-ustar);
	FR[0]+=(fact*1.                                                                      -UC[1].d )*SL;
	FR[1]+=(fact*WC[1].u                                                                 -UC[1].du)*SL;
	FR[2]+=(fact*WC[1].v                                                                 -UC[1].dv)*SL;
	FR[3]+=(fact*ustar                                                                   -UC[1].dw)*SL;
	FR[4]+=(fact*(UC[1].E/UC[1].d+(ustar-WC[1].w)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].w))))-UC[1].E )*SL;

	Us.d =(fact*1.);
	Us.du=(fact*WC[1].u);
	Us.dv=(fact*WC[1].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UC[1].E/UC[1].d+(ustar-WC[1].w)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].w)))));

#ifdef WRADHYD
	FR[6]+=(fact*WC[1].dX/WC[1].d                                                                 -UC[1].dX)*SL;
#ifdef HELIUM
	FR[7]+=(fact*WC[1].dXHE/WC[1].d                                                                 -UC[1].dXHE)*SL;
	FR[8]+=(fact*WC[1].dXXHE/WC[1].d                                                                 -UC[1].dXXHE)*SL;
#endif
#endif
      }
      else if((ustar<=0.)&&(SR>0.)){
	dgetflux_Z(&UN[1],FR);
	fact=WN[1].d*(SR-WN[1].w)/(SR-ustar);
	FR[0]+=(fact*1.                                                                      -UN[1].d )*SR;
	FR[1]+=(fact*WN[1].u                                                                 -UN[1].du)*SR;
	FR[2]+=(fact*WN[1].v                                                                 -UN[1].dv)*SR;
	FR[3]+=(fact*ustar                                                                   -UN[1].dw)*SR;
	FR[4]+=(fact*(UN[1].E/UN[1].d+(ustar-WN[1].w)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].w))))-UN[1].E )*SR;

	Us.d =(fact*1.);
	Us.du=(fact*WN[1].u);
	Us.dv=(fact*WN[1].v);
	Us.dw=(fact*ustar);
	Us.E =(fact*(UN[1].E/UN[1].d+(ustar-WN[1].w)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].w)))));

#ifdef WRADHYD
	FR[6]+=(fact*WN[1].dX/WN[1].d                                                                 -UN[1].dX)*SR;
#ifdef HELIUM
	FR[7]+=(fact*WN[1].dXHE/WN[1].d                                                                 -UN[1].dXHE)*SR;
	FR[8]+=(fact*WN[1].dXXHE/WN[1].d                                                                 -UN[1].dXXHE)*SR;
#endif

#endif
      }

      ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
      divuloc=(GAMMA-1.)*(Us.dw/Us.d)*eold;
      FR[5]=(Us.dw/Us.d*ebar);
      divu+= divuloc;


#endif

#endif
      //========================= copy the fluxes

      // Cancelling the fluxes from splitted neighbours

      for(iface=0;iface<NVAR;iface++) FL[iface]*=ffact[0]; 
      for(iface=0;iface<NVAR;iface++) FR[iface]*=ffact[1]; 


      memcpy(stencil[i].New.cell[icell].flux+4*NVAR,FL,sizeof(REAL)*NVAR);
      memcpy(stencil[i].New.cell[icell].flux+5*NVAR,FR,sizeof(REAL)*NVAR);

      stencil[i].New.cell[icell].divu=divu;

      //ready for the next oct
  }
  }
}




//============================================================================
// =============================================================================================

__global__ void dhydroM_sweepY(struct HGRID *stencil,int nread,REAL dx, REAL dt){

  int inei,icell,iface;
  int i;
  int vnei[6],vcell[6];
  
  REAL FL[NVAR],FR[NVAR];
  struct Utype Uold;
  struct Wtype Wold;
  REAL pstar,ustar;

  struct Wtype WT[6]; // FOR MUSCL RECONSTRUCTION
  struct Wtype WC[6]; // FOR MUSCL RECONSTRUCTION

  struct Utype UC[2];
  struct Utype UN[2];
  struct Wtype WN[2];

  int ioct[7]={12,14,10,16,4,22,13};
  int idxnei[6]={1,0,3,2,5,4};

  struct Wtype *curcell;

  REAL SL,SR;
  
  int ffact[2]={0,0};
  REAL fact;

  struct Utype Us;
  REAL ebar;
  REAL divu,divuloc;

  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
	
  i=bx*blockDim.x+tx;
  if(i<nread){
  for(icell=0;icell<8;icell++){ // we scan the cells
    getcellnei_gpu_hydro(icell, vnei, vcell); // we get the neighbors
      
  
      
      memset(FL,0,sizeof(REAL)*NVAR);
      memset(FR,0,sizeof(REAL)*NVAR);

      // Getting the original state ===========================
      
      curcell=&(stencil[i].oct[ioct[6]].cell[icell].field);

      divu=stencil[i].New.cell[icell].divu;

      Wold.d=curcell->d;
      Wold.u=curcell->u;
      Wold.v=curcell->v;
      Wold.w=curcell->w;
      Wold.p=curcell->p;
      Wold.a=SQRT(GAMMA*Wold.p/Wold.d);
/* #ifdef WRADHYD */
/*       Wold.X=curcell->X; */
/* #endif */
      dW2U(&Wold,&Uold); // primitive -> conservative

      REAL eold=Uold.eint;

      /* // MUSCL STATE RECONSTRUCTION */
      memset(ffact,0,sizeof(int)*2);

      dMUSCL_BOUND2(stencil+i, 13, icell, WC,dt,dx);// central

      for(iface=0;iface<2;iface++){
	inei=iface+2;
	memcpy(WC+iface,WC+inei,sizeof(struct Wtype)); // moving the data towards idx=0,1
	//memcpy(WC+iface,&Wold,sizeof(struct Wtype)); // moving the data towards idx=0,1

	dW2U(WC+iface,UC+iface);
      }

      // Neighbor MUSCL reconstruction
      for(iface=0;iface<2;iface++){
	inei=iface+2;
	dMUSCL_BOUND2(stencil+i, ioct[vnei[inei]], vcell[inei], WT,dt,dx);// 
	memcpy(WN+iface,WT+idxnei[inei],sizeof(struct Wtype)); 
	//memcpy(WN+iface,&(stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].field),sizeof(struct Wtype)); 

       	dW2U(WN+iface,UN+iface);
	
	if(!stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].split){
	  ffact[iface]=1; // we cancel the contriubtion of split neighbors
	}

      }




      // Y DIRECTION =========================================================================
      
      // --------- solving the Riemann Problems FRONT

      // Switching to Split description

/* 	// =========================================== */

#ifdef RIEMANN_HLLC
      dspeedestimateY_HLLC(&WN[0],&WC[0],&SL,&SR,&pstar,&ustar);

	if(SL>=0.){
	  dgetflux_Y(&UN[0],FL);
	  memcpy(&Us,&UN[0],sizeof(struct Utype));

	}
	else if(SR<=0.){
	  dgetflux_Y(&UC[0],FL);
	  memcpy(&Us,&UC[0],sizeof(struct Utype));
	}
	else if((SL<0.)&&(ustar>=0.)){
	  dgetflux_Y(&UN[0],FL);
	  fact=WN[0].d*(SL-WN[0].v)/(SL-ustar);
	  FL[0]+=(fact*1.                                                                      -UN[0].d )*SL;
	  FL[1]+=(fact*WN[0].u                                                                 -UN[0].du)*SL;
	  FL[2]+=(fact*ustar                                                                   -UN[0].dv)*SL;
	  FL[3]+=(fact*WN[0].w                                                                 -UN[0].dw)*SL;
	  FL[4]+=(fact*(UN[0].E/UN[0].d+(ustar-WN[0].v)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].v))))-UN[0].E )*SL;

	  Us.d =(fact*1.);
	  Us.du=(fact*WN[0].u);
	  Us.dv=(fact*ustar);
	  Us.dw=(fact*WN[0].w);
	  Us.E =(fact*(UN[0].E/UN[0].d+(ustar-WN[0].v)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].v)))));

#ifdef WRADHYD
	  FL[6]+=(fact*WN[0].dX/WN[0].d                                                                 -UN[0].dX)*SL;
#ifdef HELIUM
	  FL[7]+=(fact*WN[0].dXHE/WN[0].d                                                                 -UN[0].dXHE)*SL;
	  FL[8]+=(fact*WN[0].dXXHE/WN[0].d                                                                 -UN[0].dXXHE)*SL;
#endif

#endif

	}
	else if((ustar<=0.)&&(SR>0.)){
	  dgetflux_Y(&UC[0],FL);
	  fact=WC[0].d*(SR-WC[0].v)/(SR-ustar);
	  FL[0]+=(fact*1.                                                                      -UC[0].d )*SR;
	  FL[1]+=(fact*WC[0].u                                                                 -UC[0].du)*SR;
	  FL[2]+=(fact*ustar                                                                   -UC[0].dv)*SR;
	  FL[3]+=(fact*WC[0].w                                                                 -UC[0].dw)*SR;
	  FL[4]+=(fact*(UC[0].E/UC[0].d+(ustar-WC[0].v)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].v))))-UC[0].E )*SR;

	  Us.d =(fact*1.);
	  Us.du=(fact*WC[0].u);
	  Us.dv=(fact*ustar);
	  Us.dw=(fact*WC[0].w);
	  Us.E =(fact*(UC[0].E/UC[0].d+(ustar-WC[0].v)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].v)))));

#ifdef WRADHYD
	  FL[6]+=(fact*WC[0].dX/WC[0].d                                                                 -UC[0].dX)*SR;
#ifdef HELIUM
	  FL[7]+=(fact*WC[0].dXHE/WC[0].d                                                                 -UC[0].dXHE)*SR;
	  FL[8]+=(fact*WC[0].dXXHE/WC[0].d                                                                 -UC[0].dXXHE)*SR;
#endif
#endif

	}


	ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
	FL[5]=(Us.dv/Us.d*ebar);
	divuloc=(GAMMA-1.)*(Us.dv/Us.d)*eold;
	divu+=-divuloc;


#endif
	// ===========================================




      // --------- solving the Riemann Problems BACK


      // Switching to Split description

	//=====================================================

#ifdef RIEMANN_HLLC
	dspeedestimateY_HLLC(&WC[1],&WN[1],&SL,&SR,&pstar,&ustar);

	if(SL>=0.){
	  dgetflux_Y(&UC[1],FR);
	  memcpy(&Us,&UC[1],sizeof(struct Utype));

	}
	else if(SR<=0.){
	  dgetflux_Y(&UN[1],FR);
	  memcpy(&Us,&UN[1],sizeof(struct Utype));

	}
	else if((SL<0.)&&(ustar>=0.)){
	  dgetflux_Y(&UC[1],FR);
	  fact=WC[1].d*(SL-WC[1].v)/(SL-ustar);
	  FR[0]+=(fact*1.                                                                      -UC[1].d )*SL;
	  FR[1]+=(fact*WC[1].u                                                                 -UC[1].du)*SL;
	  FR[2]+=(fact*ustar                                                                   -UC[1].dv)*SL;
	  FR[3]+=(fact*WC[1].w                                                                 -UC[1].dw)*SL;
	  FR[4]+=(fact*(UC[1].E/UC[1].d+(ustar-WC[1].v)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].v))))-UC[1].E )*SL;

	  Us.d =(fact*1.);
	  Us.du=(fact*WC[1].u);
	  Us.dv=(fact*ustar);
	  Us.dw=(fact*WC[1].w);
	  Us.E =(fact*(UC[1].E/UC[1].d+(ustar-WC[1].v)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].v)))));

#ifdef WRADHYD
	  FR[6]+=(fact*WC[1].dX/WC[1].d                                                                 -UC[1].dX)*SL;
#ifdef HELIUM
	  FR[7]+=(fact*WC[1].dXHE/WC[1].d                                                                 -UC[1].dXHE)*SL;
	  FR[8]+=(fact*WC[1].dXXHE/WC[1].d                                                                 -UC[1].dXXHE)*SL;
#endif
#endif
	}
	else if((ustar<=0.)&&(SR>0.)){
	  dgetflux_Y(&UN[1],FR);
	  fact=WN[1].d*(SR-WN[1].v)/(SR-ustar);
	  FR[0]+=(fact*1.                                                                      -UN[1].d )*SR;
	  FR[1]+=(fact*WN[1].u                                                                 -UN[1].du)*SR;
	  FR[2]+=(fact*ustar                                                                   -UN[1].dv)*SR;
	  FR[3]+=(fact*WN[1].w                                                                 -UN[1].dw)*SR;
	  FR[4]+=(fact*(UN[1].E/UN[1].d+(ustar-WN[1].v)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].v))))-UN[1].E )*SR;

	  Us.d =(fact*1.);
	  Us.du=(fact*WN[1].u);
	  Us.dv=(fact*ustar);
	  Us.dw=(fact*WN[1].w);
	  Us.E =(fact*(UN[1].E/UN[1].d+(ustar-WN[1].v)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].v)))));
#ifdef WRADHYD
	  FR[6]+=(fact*WN[1].dX/WN[1].d                                                                 -UN[1].dX)*SR;
#ifdef HELIUM
	  FR[7]+=(fact*WN[1].dXHE/WN[1].d                                                                 -UN[1].dXHE)*SR;
	  FR[8]+=(fact*WN[1].dXXHE/WN[1].d                                                                 -UN[1].dXXHE)*SR;
#endif
#endif
	}


	ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
	divuloc=(GAMMA-1.)*(Us.dv/Us.d)*eold;
	FR[5]=(Us.dv/Us.d*ebar);
	divu+= divuloc;

#endif

      
      //========================= copy the fluxes
      // Cancelling the fluxes from splitted neighbours
	
	for(iface=0;iface<NVAR;iface++) FL[iface]*=ffact[0]; 
	for(iface=0;iface<NVAR;iface++) FR[iface]*=ffact[1]; 
	
	memcpy(stencil[i].New.cell[icell].flux+2*NVAR,FL,sizeof(REAL)*NVAR);
	memcpy(stencil[i].New.cell[icell].flux+3*NVAR,FR,sizeof(REAL)*NVAR);
	
	stencil[i].New.cell[icell].divu=divu;
	
    //ready for the next oct
  }
  }
}

//===================================================================================================
//===================================================================================================

__global__ void dhydroM_sweepX(struct HGRID *stencil, int nread,REAL dx, REAL dt){

  //printf("IN\n");
  int inei,icell,iface;
  int i;
  int vnei[6],vcell[6];

  REAL FL[NVAR],FR[NVAR];
  struct Utype Uold;
  struct Wtype Wold;
  REAL pstar,ustar;

  struct Wtype WT[6]; // FOR MUSCL RECONSTRUCTION
  struct Wtype WC[6]; // FOR MUSCL RECONSTRUCTION

  struct Utype UC[2];
  struct Utype UN[2];
  struct Wtype WN[2];

  int ioct[7]={12,14,10,16,4,22,13};
  int idxnei[6]={1,0,3,2,5,4};

  struct Wtype *curcell;

  REAL SL,SR;
  
  int ffact[2]={0,0};
  REAL fact;

  struct Utype Us;
  REAL ebar;
  REAL divu,divuloc;

  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
  
  i=bx*blockDim.x+tx;
  if(i<nread){
  for(icell=0;icell<8;icell++){ // we scan the cells
    getcellnei_gpu_hydro(icell, vnei, vcell); // we get the neighbors
      
  
      
      memset(FL,0,sizeof(REAL)*NVAR);
      memset(FR,0,sizeof(REAL)*NVAR);

      // Getting the original state ===========================
      
      curcell=&(stencil[i].oct[ioct[6]].cell[icell].field);
      
      divu=stencil[i].New.cell[icell].divu;

      Wold.d=curcell->d;
      Wold.u=curcell->u;;
      Wold.v=curcell->v;;
      Wold.w=curcell->w;;
      Wold.p=curcell->p;;
      Wold.a=SQRT(GAMMA*Wold.p/Wold.d);

      //printf("dt=%e dx=%e Old =%e %e %e %e %e\n",dt,dx,Wold.d,Wold.u,Wold.v,Wold.w,Wold.p);
/* #ifdef WRADHYD */
/*       Wold.X=curcell->X; */
/* #endif */
      dW2U(&Wold,&Uold); // primitive -> conservative
      REAL eold=Uold.eint;

      /* // MUSCL STATE RECONSTRUCTION */
      memset(ffact,0,sizeof(int)*2);

      dMUSCL_BOUND2(stencil+i, 13, icell, WC,dt,dx);// central
      for(iface=0;iface<2;iface++){
	dW2U(WC+iface,UC+iface);
      }

      // Neighbor MUSCL reconstruction
      for(iface=0;iface<2;iface++){
	inei=iface;
	dMUSCL_BOUND2(stencil+i, ioct[vnei[inei]], vcell[inei], WT,dt,dx);// 
	memcpy(WN+iface,WT+idxnei[inei],sizeof(struct Wtype)); 
       	dW2U(WN+iface,UN+iface);
	
	if(!stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].split){
	  ffact[iface]=1; // we cancel the contriubtion of split neighbors
	}

      }




      // X DIRECTION =========================================================================
      
      // --------- solving the Riemann Problems LEFT

      // Switching to Split description

/* 	// =========================================== */

#ifdef RIEMANN_HLLC
      //printf("Ho %e %e %e %e|| %e %e %e %e\n",WC[0].d,WC[0].u,WC[0].p,WC[0].a,WN[0].d,WN[0].u,WN[0].p,WN[0].a);
      dspeedestimateX_HLLC(&WN[0],&WC[0],&SL,&SR,&pstar,&ustar);
      //printf("Ha\n");

      if(SL>=0.){
	dgetflux_X(&UN[0],FL);
	memcpy(&Us,&UN[0],sizeof(struct Utype));

	}
      else if(SR<=0.){
	dgetflux_X(&UC[0],FL);
	memcpy(&Us,&UC[0],sizeof(struct Utype));
	}
      else if((SL<0.)&&(ustar>=0.)){
	dgetflux_X(&UN[0],FL);
	fact=WN[0].d*(SL-WN[0].u)/(SL-ustar);
	FL[0]+=(fact*1.                                                                      -UN[0].d )*SL;
	FL[1]+=(fact*ustar                                                                   -UN[0].du)*SL;
	FL[2]+=(fact*WN[0].v                                                                 -UN[0].dv)*SL;
	FL[3]+=(fact*WN[0].w                                                                 -UN[0].dw)*SL;
	FL[4]+=(fact*(UN[0].E/UN[0].d+(ustar-WN[0].u)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].u))))-UN[0].E )*SL;

	  Us.d =(fact*1.);
	  Us.du=(fact*ustar);
	  Us.dv=(fact*WN[0].v);
	  Us.dw=(fact*WN[0].w);
	  Us.E =(fact*(UN[0].E/UN[0].d+(ustar-WN[0].u)*(ustar+WN[0].p/(WN[0].d*(SL-WN[0].u)))));

#ifdef WRADHYD
	 FL[6]+=(fact*WN[0].dX/WN[0].d                                                                 -UN[0].dX)*SL;
#ifdef HELIUM
	 FL[7]+=(fact*WN[0].dXHE/WN[0].d                                                                 -UN[0].dXHE)*SL;
	 FL[8]+=(fact*WN[0].dXXHE/WN[0].d                                                                 -UN[0].dXXHE)*SL;
#endif
#endif
	}
      else if((ustar<=0.)&&(SR>0.)){
	dgetflux_X(&UC[0],FL);
	  fact=WC[0].d*(SR-WC[0].u)/(SR-ustar);
	  FL[0]+=(fact*1.                                                                      -UC[0].d )*SR;
	  FL[1]+=(fact*ustar                                                                   -UC[0].du)*SR;
	  FL[2]+=(fact*WC[0].v                                                                 -UC[0].dv)*SR;
	  FL[3]+=(fact*WC[0].w                                                                 -UC[0].dw)*SR;
	  FL[4]+=(fact*(UC[0].E/UC[0].d+(ustar-WC[0].u)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].u))))-UC[0].E )*SR;

	  Us.d =(fact*1.);
	  Us.du=(fact*ustar);
	  Us.dv=(fact*WC[0].v);
	  Us.dw=(fact*WC[0].w);
	  Us.E =(fact*(UC[0].E/UC[0].d+(ustar-WC[0].u)*(ustar+WC[0].p/(WC[0].d*(SR-WC[0].u)))));
#ifdef WRADHYD
	  FL[6]+=(fact*WC[0].dX/WC[0].d                                                                 -UC[0].dX)*SR;
#ifdef HELIUM
	  FL[7]+=(fact*WC[0].dXHE/WC[0].d                                                                 -UC[0].dXHE)*SR;
	  FL[8]+=(fact*WC[0].dXXHE/WC[0].d                                                                 -UC[0].dXXHE)*SR;
#endif
#endif

	}


      ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
      divuloc=(GAMMA-1.)*(Us.du/Us.d)*eold;
      FL[5]=(Us.du/Us.d*ebar);
      divu+=-divuloc;


#endif
	

	// ===========================================


      

      // --------- solving the Riemann Problems RIGHT


      // Switching to Split description

	//=====================================================

#ifdef RIEMANN_HLLC
      dspeedestimateX_HLLC(&WC[1],&WN[1],&SL,&SR,&pstar,&ustar);

	if(SL>=0.){
	  dgetflux_X(&UC[1],FR);
	  memcpy(&Us,&UC[1],sizeof(struct Utype));

	}
	else if(SR<=0.){
	  dgetflux_X(&UN[1],FR);
	  memcpy(&Us,&UN[1],sizeof(struct Utype));

	}
	else if((SL<0.)&&(ustar>=0.)){
	  dgetflux_X(&UC[1],FR);
	  fact=WC[1].d*(SL-WC[1].u)/(SL-ustar);
	  FR[0]+=(fact*1.                                                                      -UC[1].d )*SL;
	  FR[1]+=(fact*ustar                                                                   -UC[1].du)*SL;
	  FR[2]+=(fact*WC[1].v                                                                 -UC[1].dv)*SL;
	  FR[3]+=(fact*WC[1].w                                                                 -UC[1].dw)*SL;
	  FR[4]+=(fact*(UC[1].E/UC[1].d+(ustar-WC[1].u)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].u))))-UC[1].E )*SL;

	  Us.d =(fact*1.);
	  Us.du=(fact*ustar);
	  Us.dv=(fact*WC[1].v);
	  Us.dw=(fact*WC[1].w);
	  Us.E =(fact*(UC[1].E/UC[1].d+(ustar-WC[1].u)*(ustar+WC[1].p/(WC[1].d*(SL-WC[1].u)))));
#ifdef WRADHYD
	  FR[6]+=(fact*WC[1].dX/WC[1].d                                                                 -UC[1].dX)*SL;
#ifdef HELIUM
	  FR[7]+=(fact*WC[1].dXHE/WC[1].d                                                                 -UC[1].dXHE)*SL;
	  FR[8]+=(fact*WC[1].dXXHE/WC[1].d                                                                 -UC[1].dXXHE)*SL;
#endif
#endif
	}
	else if((ustar<=0.)&&(SR>0.)){
	  dgetflux_X(&UN[1],FR);
	  fact=WN[1].d*(SR-WN[1].u)/(SR-ustar);
	  FR[0]+=(fact*1.                                                                      -UN[1].d )*SR;
	  FR[1]+=(fact*ustar                                                                   -UN[1].du)*SR;
	  FR[2]+=(fact*WN[1].v                                                                 -UN[1].dv)*SR;
	  FR[3]+=(fact*WN[1].w                                                                 -UN[1].dw)*SR;
	  FR[4]+=(fact*(UN[1].E/UN[1].d+(ustar-WN[1].u)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].u))))-UN[1].E )*SR;

	  Us.d =(fact*1.);
	  Us.du=(fact*ustar);
	  Us.dv=(fact*WN[1].v);
	  Us.dw=(fact*WN[1].w);
	  Us.E =(fact*(UN[1].E/UN[1].d+(ustar-WN[1].u)*(ustar+WN[1].p/(WN[1].d*(SR-WN[1].u)))));
#ifdef WRADHYD
	  FR[6]+=(fact*WN[1].dX/WN[1].d                                                                 -UN[1].dX)*SR;
#ifdef HELIUM
	  FR[7]+=(fact*WN[1].dXHE/WN[1].d                                                                 -UN[1].dXHE)*SR;
	  FR[8]+=(fact*WN[1].dXXHE/WN[1].d                                                                 -UN[1].dXXHE)*SR;
#endif
#endif
	}

      ebar=(Us.E-0.5*(Us.du*Us.du+Us.dv*Us.dv+Us.dw*Us.dw)/Us.d); 
      divuloc=(GAMMA-1.)*(Us.du/Us.d)*eold;
      FR[5]=(Us.du/Us.d*ebar);
      divu+= divuloc;

#endif





      
      //========================= copy the fluxes
      // Cancelling the fluxes from splitted neighbours

      for(iface=0;iface<NVAR;iface++) FL[iface]*=ffact[0]; 
      for(iface=0;iface<NVAR;iface++) FR[iface]*=ffact[1]; 

      memcpy(stencil[i].New.cell[icell].flux+0*NVAR,FL,sizeof(REAL)*NVAR);
      memcpy(stencil[i].New.cell[icell].flux+1*NVAR,FR,sizeof(REAL)*NVAR);

      stencil[i].New.cell[icell].divu=divu;

    //ready for the next oct
  }
  }
}


// ==============================================================================================================
// ==============================================================================================================

__global__ void dupdatefield(struct HGRID *stencil, int nread, int stride, struct CPUINFO *cpu, REAL dxcur, REAL dtnew)
{
  int i,icell;
  struct Utype U;
  REAL one;
  int flx;
  REAL dtsurdx=dtnew/dxcur;
  REAL F[NFLUX];


  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
  i=bx*blockDim.x+tx;
  if(i<nread){
  for(icell=0;icell<8;icell++){ // we scan the cells
      
    if(stencil[i].oct[13].cell[icell].split) continue;
    memcpy(F,stencil[i].New.cell[icell].flux,sizeof(REAL)*NFLUX);// New fluxes from the stencil

    // ==== updating
    // actually we compute and store the delta U only
    one=1.;
    memset(&U,0,sizeof(struct Utype)); // setting delta U
    for(flx=0;flx<6;flx++){
      U.d +=F[0+flx*NVAR]*dtsurdx*one;
      U.du+=F[1+flx*NVAR]*dtsurdx*one;
      U.dv+=F[2+flx*NVAR]*dtsurdx*one;
      U.dw+=F[3+flx*NVAR]*dtsurdx*one;
      U.E +=F[4+flx*NVAR]*dtsurdx*one;
      U.eint+=F[5+flx*NVAR]*dtsurdx*one;
      
#ifdef WRADHYD
#ifndef NOADX
      U.dX+=F[6+flx*NVAR]*dtsurdx*one;
#ifdef HELIUM
	U.dXHE+=F[7+flx*NVAR]*dtsurdx*one;
	U.dXXHE+=F[8+flx*NVAR]*dtsurdx*one;
#endif

#else
      U.dX+=0.;
#ifdef HELIUM
	U.dXHE+=0.;
	U.dXXHE+=0.;
#endif
#endif
#endif
      one*=-1.;
    }
    // scatter back the delta Uwithin the stencil
    
    memcpy(&(stencil[i].New.cell[icell].deltaU),&U,sizeof(struct Utype));

  }
  }
}

// =======================================================

//=======================================================================
//=======================================================================

int advancehydroGPU(struct OCT **firstoct, int level, struct CPUINFO *cpu, struct HGRID *stencil, int stride, REAL dxcur, REAL dtnew){

  struct OCT *nextoct;
  struct OCT *curoct;
  struct OCT *curoct0;
  int nreadtot,nread;
  /* REAL t[10]; */
  /* REAL tg=0.,th=0.,tu=0.,ts=0.; */
  int is;
  int offset;
  CUDA_CHECK_ERROR("Hydro start");
  
  // --------------- setting the first oct of the level
  nextoct=firstoct[level-1];
  nreadtot=0;
  int ng;
  int nt;

  cudaStream_t stream[cpu->nstream]; // Not fully regular expression
  int vnread[cpu->nstream];
  // creating the streams
  for(is=0;is<cpu->nstream;is++){
    cudaStreamCreate(&stream[is]);
  }
  
  // Calculations

  if((nextoct!=NULL)&&(cpu->noct[level-1]!=0)){
    do {
      curoct0=nextoct;
      curoct=curoct0;
      
      //t[0]=MPI_Wtime();
  
#if 1
      offset=0;
      // streaming ====================
      for(is=0;is<cpu->nstream;is++){

	// ------------ gathering the stencil value values
	//printf("offser=%d\n",offset);
	curoct=nextoct;
	if(curoct!=NULL){
	  nextoct= gatherstencil(curoct,stencil+offset,stride/cpu->nstream,cpu, vnread+is);
	  if(vnread[is]!=0){
	    ng=((vnread[is]-1)/cpu->nthread)+1; // +1 is for leftovers
	    
	    if(ng==1){
	      nt=vnread[is];
	    }
	    else{
	      nt=cpu->nthread;
	    }
	    
	  dim3 gridoct(ng);
	  dim3 blockoct(nt);
	  
	  
#ifndef NOCOMP     
	  cudaMemcpyAsync(cpu->hyd_stencil+offset,stencil+offset,vnread[is]*sizeof(struct HGRID),cudaMemcpyHostToDevice,stream[is]);  
	  
	  //printf("Sweep hydro dt=%e dx=%e\n",dtnew,dxcur);
	  //CUDA_CHECK_ERROR("Sweep hydro");
	  // ------------ solving the hydro

	  dhydroM_sweepX<<<gridoct,blockoct,0,stream[is]>>>(cpu->hyd_stencil+offset,vnread[is],dxcur,dtnew);
	  dhydroM_sweepY<<<gridoct,blockoct,0,stream[is]>>>(cpu->hyd_stencil+offset,vnread[is],dxcur,dtnew);   
   	  dhydroM_sweepZ<<<gridoct,blockoct,0,stream[is]>>>(cpu->hyd_stencil+offset,vnread[is],dxcur,dtnew);     
	  
	  //printf("Sweep hydro stop\n");
	  // ------------ updating values within the stencil

	  dupdatefield<<<gridoct,blockoct,0,stream[is]>>>(cpu->hyd_stencil+offset,vnread[is],stride,cpu,dxcur,dtnew);
	
	  cudaMemcpyAsync(stencil+offset,cpu->hyd_stencil+offset,vnread[is]*sizeof(struct HGRID),cudaMemcpyDeviceToHost,stream[is]);  
#endif
	
	  offset+=vnread[is];
	  }
	}
      }
#endif

      
	/* dev_updatefield<<<gridoct2,blockoct2>>>(cpu->hyd_stencil,nread,stride,cpu,dxcur,dtnew); */
      
      cudaDeviceSynchronize();
	// ------------ scatter back the FLUXES
	//cudaMemcpy(stencil,cpu->hyd_stencil,nread*sizeof(struct HGRID),cudaMemcpyDeviceToHost);  
	nread=offset;
	nextoct=scatterstencil(curoct0,stencil, nread, cpu,dxcur,dtnew);
      
	//t[8]=MPI_Wtime();
      
	nreadtot+=nread;
      
      
	/* ts+=(t[8]-t[6]); */
	/* tu+=(t[6]-t[4]); */
	/* th+=(t[4]-t[2]); */
	/* tg+=(t[2]-t[0]); */
	//printf("Start Error Hyd =%s nreadtot=%d\n",cudaGetErrorString(cudaGetLastError()),nreadtot);
      }while(nextoct!=NULL);
    
  }
  //printf("GPU | tgat=%e tcal=%e tup=%e tscat=%e\n",tg,th,tu,ts);

  // Destroying the streams
  for(is=0;is<cpu->nstream;is++){
    cudaStreamDestroy(stream[is]);
  }

 // printf("Start Error Hyd =%s nreadtot=%d\n",cudaGetErrorString(cudaGetLastError()),nreadtot);
  CUDA_CHECK_ERROR("Hydro Stop");

  return nreadtot;
}

#endif
