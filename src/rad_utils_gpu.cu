
#include <hip/hip_runtime.h>
				       
#ifdef WRAD
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "prototypes.h"
#include "oct.h"
#include <string.h>

#include <mpi.h>

#ifdef WCHEM
#include "chem_utils.cuh"
#endif

#include "gpu_type.h"

extern "C" struct OCT *gatherstencilrad(struct OCT *octstart, struct RGRID *stencil, int stride, struct CPUINFO *cpu, int *nread, REAL cloc);
extern "C" struct OCT *scatterstencilrad(struct OCT *octstart, struct RGRID *stencil, int stride, struct CPUINFO *cpu, REAL dxcur, REAL dtnew, REAL cloc);
extern "C" int advanceradGPU (struct OCT **firstoct, int level, struct CPUINFO *cpu, struct RGRID *stencil, int stride, REAL dxcur, REAL dtnew,REAL aexp, struct RUNPARAMS *param, int chemonly);


extern "C" void create_radstencil_GPU(struct CPUINFO *cpu, int stride);
extern "C" void create_pinned_stencil_rad(struct RGRID **stencil, int stride);
extern "C" void destroy_radstencil_GPU(struct CPUINFO *cpu, int stride);
extern "C" void destroy_pinned_stencil_rad(struct RGRID **stencil, int stride);
extern "C" void create_param_GPU(struct RUNPARAMS *param, struct CPUINFO *cpu);

// ===================================================================
void create_radstencil_GPU(struct CPUINFO *cpu, int stride){
  cudaMalloc((void **)&(cpu->rad_stencil),sizeof(struct RGRID)*stride);
  cudaDeviceSynchronize(); 
  //  printf("Start Error =%s\n",cudaGetErrorString(cudaGetLastError()));

  
}

// ===================================================================
void create_pinned_stencil_rad(struct RGRID **stencil, int stride){
  cudaMallocHost( (void**)stencil, sizeof(struct RGRID)*stride );
  cudaDeviceSynchronize(); 
  //  printf("Start2 Error =%s\n",cudaGetErrorString(cudaGetLastError()));
}

// ===================================================================
void destroy_radstencil_GPU(struct CPUINFO *cpu, int stride){
  cudaFree(cpu->rad_stencil);
}

// ===================================================================
void destroy_pinned_stencil_rad(struct RGRID **stencil, int stride){
  cudaFreeHost(stencil);
}


// ===================================================================
void create_param_GPU(struct RUNPARAMS *param, struct CPUINFO *cpu){

  struct RUNPARAMS paramgpu;
  memcpy(&paramgpu,param,sizeof(struct RUNPARAMS));

  struct COSMOPARAM CG;
  struct COSMOPARAM *dCG;
  memcpy(&CG,param->cosmo,sizeof(struct COSMOPARAM));

  cudaMalloc((void **)&(dCG),sizeof(struct COSMOPARAM)); // allocate cosmo struct on GPU
  cudaMemcpy(dCG,&CG,sizeof(struct COSMOPARAM),cudaMemcpyHostToDevice);  
  paramgpu.cosmo=dCG; // we save the pointer location

#ifdef WRAD
  REAL *space_bound;
  cudaMalloc((void **)&(space_bound),(param->atomic.ngrp_space+1)*sizeof(REAL)); // allocate cosmo struct on GPU
  cudaMemcpy(space_bound,param->atomic.space_bound,(param->atomic.ngrp_space+1)*sizeof(REAL),cudaMemcpyHostToDevice);
  paramgpu.atomic.space_bound=space_bound;

  REAL *time_bound;
  cudaMalloc((void **)&(time_bound),(param->atomic.ngrp_time+1)*sizeof(REAL)); // allocate cosmo struct on GPU
  cudaMemcpy(time_bound,param->atomic.time_bound,(param->atomic.ngrp_time+1)*sizeof(REAL),cudaMemcpyHostToDevice);
  paramgpu.atomic.time_bound=time_bound;

  REAL *hnu;
  cudaMalloc((void **)&(hnu),(param->atomic.n)*sizeof(REAL)); // allocate cosmo struct on GPU
  cudaMemcpy(hnu,param->atomic.hnu,(param->atomic.n)*sizeof(REAL),cudaMemcpyHostToDevice);
  paramgpu.atomic.hnu=hnu;

  REAL *alphae;
  cudaMalloc((void **)&(alphae),(param->atomic.n)*sizeof(REAL)); // allocate cosmo struct on GPU
  cudaMemcpy(alphae,param->atomic.alphae,(param->atomic.n)*sizeof(REAL),cudaMemcpyHostToDevice);
  paramgpu.atomic.alphae=alphae;

  REAL *alphai;
  cudaMalloc((void **)&(alphai),(param->atomic.n)*sizeof(REAL)); // allocate cosmo struct on GPU
  cudaMemcpy(alphai,param->atomic.alphai,(param->atomic.n)*sizeof(REAL),cudaMemcpyHostToDevice);
  paramgpu.atomic.alphai=alphai;

  REAL *factgrp;
  cudaMalloc((void **)&(factgrp),(param->atomic.n)*sizeof(REAL)); // allocate cosmo struct on GPU
  cudaMemcpy(factgrp,param->atomic.factgrp,(param->atomic.n)*sizeof(REAL),cudaMemcpyHostToDevice);
  paramgpu.atomic.factgrp=factgrp;
#endif


  cudaMalloc((void **)&(cpu->dparam),sizeof(struct RUNPARAMS));
  cudaMemcpy(cpu->dparam,&paramgpu,sizeof(struct RUNPARAMS),cudaMemcpyHostToDevice);  
  cudaDeviceSynchronize(); 
}


// =======================================================
__device__ void getcellnei_gpu_rad(int cindex, int *neip, int *cell)
{
  switch(cindex){
  case 0:
    neip[0]=0;cell[0]=1;
    neip[1]=6;cell[1]=1;
    neip[2]=2;cell[2]=2;
    neip[3]=6;cell[3]=2;
    neip[4]=4;cell[4]=4;
    neip[5]=6;cell[5]=4;
    break;
  case 1:
    neip[0]=6;cell[0]=0;
    neip[1]=1;cell[1]=0;
    neip[2]=2;cell[2]=3;
    neip[3]=6;cell[3]=3;
    neip[4]=4;cell[4]=5;
    neip[5]=6;cell[5]=5;
    break;
  case 2:
    neip[0]=0;cell[0]=3;
    neip[1]=6;cell[1]=3;
    neip[2]=6;cell[2]=0;
    neip[3]=3;cell[3]=0;
    neip[4]=4;cell[4]=6;
    neip[5]=6;cell[5]=6;
    break;
  case 3:
    neip[0]=6;cell[0]=2;
    neip[1]=1;cell[1]=2;
    neip[2]=6;cell[2]=1;
    neip[3]=3;cell[3]=1;
    neip[4]=4;cell[4]=7;
    neip[5]=6;cell[5]=7;
    break;
  case 4:
    neip[0]=0;cell[0]=5;
    neip[1]=6;cell[1]=5;
    neip[2]=2;cell[2]=6;
    neip[3]=6;cell[3]=6;
    neip[4]=6;cell[4]=0;
    neip[5]=5;cell[5]=0;
    break;
  case 5:
    neip[0]=6;cell[0]=4;
    neip[1]=1;cell[1]=4;
    neip[2]=2;cell[2]=7;
    neip[3]=6;cell[3]=7;
    neip[4]=6;cell[4]=1;
    neip[5]=5;cell[5]=1;
    break;
  case 6:
    neip[0]=0;cell[0]=7;
    neip[1]=6;cell[1]=7;
    neip[2]=6;cell[2]=4;
    neip[3]=3;cell[3]=4;
    neip[4]=6;cell[4]=2;
    neip[5]=5;cell[5]=2;
    break;
  case 7:
    neip[0]=6;cell[0]=6;
    neip[1]=1;cell[1]=6;
    neip[2]=6;cell[2]=5;
    neip[3]=3;cell[3]=5;
    neip[4]=6;cell[4]=3;
    neip[5]=5;cell[5]=3;
    break;
  }

}


//================================================================================
__device__ void ddiffR(struct Rtype *W2, struct Rtype *W1, struct Rtype *WR){
  int igrp;
  for(igrp=0;igrp<NGRP;igrp++){
    WR->e[igrp]=W2->e[igrp]- W1->e[igrp];
    WR->fx[igrp]=W2->fx[igrp]- W1->fx[igrp];
    WR->fy[igrp]=W2->fy[igrp]- W1->fy[igrp];
    WR->fz[igrp]=W2->fz[igrp]- W1->fz[igrp];
    WR->src[igrp]=W2->src[igrp]- W1->src[igrp];
  }

#ifdef SUPERNOVAE
    WR->snfb=W2->snfb- W1->snfb;
#endif
#ifdef WCHEM
    WR->nhplus=W2->nhplus-W1->nhplus;
#ifdef HELIUM
    WR->nheplus=W2->nheplus-W1->nheplus;
    WR->nhepplus=W2->nhepplus-W1->nhepplus;
#endif
    WR->eint=W2->eint-W1->eint;
    WR->nh=W2->nh-W1->nh;
#endif
}




//================================================================================
__device__ void dminmod_R(struct Rtype *Wm, struct Rtype *Wp, struct Rtype *Wr){

  REAL beta=1.; // 1. for MINBEE 2. for SUPERBEE
  // FLUX LIMITER
  int igrp;
  for(igrp=0;igrp<NGRP;igrp++){

    if(Wp->e[igrp]>0){
      Wr->e[igrp]=FMAX(FMAX(0.,FMIN(beta*Wm->e[igrp],Wp->e[igrp])),FMIN(Wm->e[igrp],beta*Wp->e[igrp]));
    }
    else{
      Wr->e[igrp]=FMIN(FMIN(0.,FMAX(beta*Wm->e[igrp],Wp->e[igrp])),FMAX(Wm->e[igrp],beta*Wp->e[igrp]));
    }


    if(Wp->fx[igrp]>0){
      Wr->fx[igrp]=FMAX(FMAX(0.,FMIN(beta*Wm->fx[igrp],Wp->fx[igrp])),FMIN(Wm->fx[igrp],beta*Wp->fx[igrp]));
    }
    else{
      Wr->fx[igrp]=FMIN(FMIN(0.,FMAX(beta*Wm->fx[igrp],Wp->fx[igrp])),FMAX(Wm->fx[igrp],beta*Wp->fx[igrp]));
    }


    if(Wp->fy[igrp]>0){
      Wr->fy[igrp]=FMAX(FMAX(0.,FMIN(beta*Wm->fy[igrp],Wp->fy[igrp])),FMIN(Wm->fy[igrp],beta*Wp->fy[igrp]));
    }
    else{
      Wr->fy[igrp]=FMIN(FMIN(0.,FMAX(beta*Wm->fy[igrp],Wp->fy[igrp])),FMAX(Wm->fy[igrp],beta*Wp->fy[igrp]));
    }


    if(Wp->fz[igrp]>0){
      Wr->fz[igrp]=FMAX(FMAX(0.,FMIN(beta*Wm->fz[igrp],Wp->fz[igrp])),FMIN(Wm->fz[igrp],beta*Wp->fz[igrp]));
    }
    else{
      Wr->fz[igrp]=FMIN(FMIN(0.,FMAX(beta*Wm->fz[igrp],Wp->fz[igrp])),FMAX(Wm->fz[igrp],beta*Wp->fz[igrp]));
    }

    if(Wp->src[igrp]>0){
      Wr->src[igrp]=FMAX(FMAX(0.,FMIN(beta*Wm->src[igrp],Wp->src[igrp])),FMIN(Wm->src[igrp],beta*Wp->src[igrp]));
    }
    else{
      Wr->src[igrp]=FMIN(FMIN(0.,FMAX(beta*Wm->src[igrp],Wp->src[igrp])),FMAX(Wm->src[igrp],beta*Wp->src[igrp]));
    }

  }




#ifdef SUPERNOVAE
  if(Wp->snfb>0){
      Wr->snfb=FMAX(FMAX(0.,FMIN(beta*Wm->snfb,Wp->snfb)),FMIN(Wm->snfb,beta*Wp->snfb));
    }
    else{
      Wr->snfb=FMIN(FMIN(0.,FMAX(beta*Wm->snfb,Wp->snfb)),FMAX(Wm->snfb,beta*Wp->snfb));
    }
#endif

#ifdef WCHEM
  if(Wp->nhplus>0){
    Wr->nhplus=FMAX(FMAX(0.,FMIN(beta*Wm->nhplus,Wp->nhplus)),FMIN(Wm->nhplus,beta*Wp->nhplus));
  }
  else{
    Wr->nhplus=FMIN(FMIN(0.,FMAX(beta*Wm->nhplus,Wp->nhplus)),FMAX(Wm->nhplus,beta*Wp->nhplus));
  }

#ifdef HELIUM
  if(Wp->nheplus>0){
    Wr->nheplus=FMAX(FMAX(0.,FMIN(beta*Wm->nheplus,Wp->nheplus)),FMIN(Wm->nheplus,beta*Wp->nheplus));
  }
  else{
    Wr->nheplus=FMIN(FMIN(0.,FMAX(beta*Wm->nheplus,Wp->nheplus)),FMAX(Wm->nheplus,beta*Wp->nheplus));
  }

  if(Wp->nhepplus>0){
    Wr->nhepplus=FMAX(FMAX(0.,FMIN(beta*Wm->nhepplus,Wp->nhepplus)),FMIN(Wm->nhepplus,beta*Wp->nhepplus));
  }
  else{
    Wr->nhepplus=FMIN(FMIN(0.,FMAX(beta*Wm->nhepplus,Wp->nhepplus)),FMAX(Wm->nhepplus,beta*Wp->nhepplus));
  }

#endif

  if(Wp->eint>0){
    Wr->eint=FMAX(FMAX(0.,FMIN(beta*Wm->eint,Wp->eint)),FMIN(Wm->eint,beta*Wp->eint));
  }
  else{
    Wr->eint=FMIN(FMIN(0.,FMAX(beta*Wm->eint,Wp->eint)),FMAX(Wm->eint,beta*Wp->eint));
  }

  if(Wp->nh>0){
    Wr->nh=FMAX(FMAX(0.,FMIN(beta*Wm->nh,Wp->nh)),FMIN(Wm->nh,beta*Wp->nh));
  }
  else{
    Wr->nh=FMIN(FMIN(0.,FMAX(beta*Wm->nh,Wp->nh)),FMAX(Wm->nh,beta*Wp->nh));
  }
#endif

}



//================================================================================
__device__ void dinterpminmod_R(struct Rtype *W0, struct Rtype *Wp, struct Rtype *Dx, struct Rtype *Dy, struct Rtype *Dz,REAL dx,REAL dy,REAL dz){
  int igrp;
  for(igrp=0;igrp<NGRP;igrp++){
    Wp->e[igrp] =W0->e[igrp] +dx*Dx->e[igrp] +dy*Dy->e[igrp] +dz*Dz->e[igrp];
    Wp->fx[igrp] =W0->fx[igrp] +dx*Dx->fx[igrp] +dy*Dy->fx[igrp] +dz*Dz->fx[igrp];
    Wp->fy[igrp] =W0->fy[igrp] +dx*Dx->fy[igrp] +dy*Dy->fy[igrp] +dz*Dz->fy[igrp];
    Wp->fz[igrp] =W0->fz[igrp] +dx*Dx->fz[igrp] +dy*Dy->fz[igrp] +dz*Dz->fz[igrp];
    Wp->src[igrp] =W0->src[igrp] +dx*Dx->src[igrp] +dy*Dy->src[igrp] +dz*Dz->src[igrp];
  }


#ifdef WCHEM
    Wp->nhplus =W0->nhplus + dx*Dx->nhplus + dy*Dy->nhplus + dz*Dz->nhplus;
#ifdef HELIUM
    Wp->nheplus =W0->nheplus + dx*Dx->nheplus + dy*Dy->nheplus + dz*Dz->nheplus;
    Wp->nhepplus =W0->nhepplus + dx*Dx->nhepplus + dy*Dy->nhepplus + dz*Dz->nhepplus;
#endif
    Wp->eint =W0->eint +dx*Dx->eint +dy*Dy->eint +dz*Dz->eint;
    Wp->nh =W0->nh +dx*Dx->nh +dy*Dy->nh +dz*Dz->nh;
#endif
#ifdef SUPERNOVAE
    Wp->snfb =W0->snfb +dx*Dx->snfb +dy*Dy->snfb +dz*Dz->snfb;
#endif
}



//================================================================================
//================================================================================
__device__ REAL dEddington(REAL fx, REAL fy, REAL fz, REAL ee, REAL c,int i,int j)
{
  REAL c2e=ee*c*c; // 2 flop
  REAL ff=0.;
  REAL arg,chi,res=0.;
  REAL n[3];
  n[0]=0.;n[1]=0.;n[2]=0.;

  if(ee>0)
    {
      ff=SQRT(fx*fx+fy*fy+fz*fz); // 6 flop
      if(ff>0)
	{
	  n[0]=fx/ff; 
	  n[1]=fy/ff;
	  n[2]=fz/ff; 
	}
      ff=ff/(c*ee); // 2flop
    }
  
  arg=FMAX(4.-3.*ff*ff,0.); // 4 flop
  chi=(3.+4.*ff*ff)/(5.+2.*SQRT(arg)); // 7 flops

  if(i==j) res=(1.-chi)/2.*c2e; // 1 flops on average
  arg=(3.*chi-1.)/2.*c2e;
  res+=arg*n[i]*n[j];

  return res;
}



// =============================================================================================================

__global__ void drad_sweepX(struct RGRID *stencil, int level, int curcpu, int nread,int stride,REAL dx, REAL dt, REAL c){

  int inei,icell,iface;
  int i,igrp;
  int vnei[6],vcell[6];

  REAL FL[NVAR_R*NGRP],FR[NVAR_R*NGRP];

  struct Rtype RC[2];
  struct Rtype RN[2];

  int ioct[7]={0,1,2,3,4,5,6};

  struct Rtype *curcell;

  int ffact[2]={0,0};
  REAL fp,fm;
  REAL up,um;

  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
  
  i=bx*blockDim.x+tx;
  if(i<nread){
  for(icell=0;icell<8;icell++){ // we scan the cells
    getcellnei_gpu_rad(icell, vnei, vcell); // we get the neighbors
      
    memset(FL,0,sizeof(REAL)*NVAR_R);
    memset(FR,0,sizeof(REAL)*NVAR_R);

    // Getting the original state ===========================
      
    curcell=&(stencil[i].oct[ioct[6]].cell[icell].rfield);
      
    /* // "MUSCL-LIKE" STATE RECONSTRUCTION */
    memset(ffact,0,sizeof(int)*2);
    for(iface=0;iface<2;iface++){
      memcpy(RC+iface,curcell,sizeof(struct Rtype));
    }

    // Neighbor "MUSCL-LIKE" reconstruction
    for(iface=0;iface<2;iface++){
      inei=iface;
      memcpy(RN+iface,&(stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].rfield),sizeof(struct Rtype));

      int condsplit;
#ifdef COARSERAD
      condsplit=1;
#else
      condsplit=(!stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].split);
#endif
      if(condsplit){
	ffact[iface]=1; // we cancel the contriubtion of split neighbors
      }
    }

    // X DIRECTION =========================================================================
      
    // --------- solving the Riemann Problems LEFT

    for(igrp=0;igrp<NGRP;igrp++){
	
      // E
      up=RC[0].e[igrp];
      um=RN[0].e[igrp];

      fp=RC[0].fx[igrp];
      fm=RN[0].fx[igrp];

      FL[0+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);
	
      //FX

      up=RC[0].fx[igrp];
      um=RN[0].fx[igrp];

      fp=dEddington(RC[0].fx[igrp],RC[0].fy[igrp],RC[0].fz[igrp],RC[0].e[igrp],c,0,0);
      fm=dEddington(RN[0].fx[igrp],RN[0].fy[igrp],RN[0].fz[igrp],RN[0].e[igrp],c,0,0);

      FL[1+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

      //FY

      up=RC[0].fy[igrp];
      um=RN[0].fy[igrp];

      fp=dEddington(RC[0].fx[igrp],RC[0].fy[igrp],RC[0].fz[igrp],RC[0].e[igrp],c,1,0);
      fm=dEddington(RN[0].fx[igrp],RN[0].fy[igrp],RN[0].fz[igrp],RN[0].e[igrp],c,1,0);

      FL[2+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

      //FZ

      up=RC[0].fz[igrp];
      um=RN[0].fz[igrp];

      fp=dEddington(RC[0].fx[igrp],RC[0].fy[igrp],RC[0].fz[igrp],RC[0].e[igrp],c,2,0);
      fm=dEddington(RN[0].fx[igrp],RN[0].fy[igrp],RN[0].fz[igrp],RN[0].e[igrp],c,2,0);

      FL[3+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

    }
      

    // ===========================================

    // --------- solving the Riemann Problems RIGHT

    for(igrp=0;igrp<NGRP;igrp++){
	
      // E
      up=RN[1].e[igrp];
      um=RC[1].e[igrp];

      fp=RN[1].fx[igrp];
      fm=RC[1].fx[igrp];

      FR[0+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);
	
      //FX

      up=RN[1].fx[igrp];
      um=RC[1].fx[igrp];

      fp=dEddington(RN[1].fx[igrp],RN[1].fy[igrp],RN[1].fz[igrp],RN[1].e[igrp],c,0,0);
      fm=dEddington(RC[1].fx[igrp],RC[1].fy[igrp],RC[1].fz[igrp],RC[1].e[igrp],c,0,0);

      FR[1+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

      //FY

      up=RN[1].fy[igrp];
      um=RC[1].fy[igrp];

      fp=dEddington(RN[1].fx[igrp],RN[1].fy[igrp],RN[1].fz[igrp],RN[1].e[igrp],c,1,0);
      fm=dEddington(RC[1].fx[igrp],RC[1].fy[igrp],RC[1].fz[igrp],RC[1].e[igrp],c,1,0);

      FR[2+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

      //FX

      up=RN[1].fz[igrp];
      um=RC[1].fz[igrp];

      fp=dEddington(RN[1].fx[igrp],RN[1].fy[igrp],RN[1].fz[igrp],RN[1].e[igrp],c,2,0);
      fm=dEddington(RC[1].fx[igrp],RC[1].fy[igrp],RC[1].fz[igrp],RC[1].e[igrp],c,2,0);

      FR[3+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

    }
      
    //========================= copy the fluxes
    // Cancelling the fluxes from splitted neighbours


    for(igrp=0;igrp<NGRP;igrp++){
      for(iface=0;iface<NVAR_R;iface++) FL[iface+igrp*NVAR_R]*=ffact[0]; 
      for(iface=0;iface<NVAR_R;iface++) FR[iface+igrp*NVAR_R]*=ffact[1]; 
    }

    memcpy(stencil[i].New.cell[icell].rflux+0*NVAR_R*NGRP,FL,sizeof(REAL)*NVAR_R*NGRP);
    memcpy(stencil[i].New.cell[icell].rflux+1*NVAR_R*NGRP,FR,sizeof(REAL)*NVAR_R*NGRP);

    //ready for the next oct
  }
}
}



// =============================================================================================================

__global__ void drad_sweepY(struct RGRID *stencil, int level, int curcpu, int nread,int stride,REAL dx, REAL dt, REAL c){

  int inei,icell,iface;
  int i,igrp;
  int vnei[6],vcell[6];

  REAL FL[NVAR_R*NGRP],FR[NVAR_R*NGRP];

  struct Rtype RC[2];
  struct Rtype RN[2];

  int ioct[7]={0,1,2,3,4,5,6};

  struct Rtype *curcell;

  int ffact[2]={0,0};
  REAL fp,fm;
  REAL up,um;
  
  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
  
  i=bx*blockDim.x+tx;
  if(i<nread){
  for(icell=0;icell<8;icell++){ // we scan the cells
    getcellnei_gpu_rad(icell, vnei, vcell); // we get the neighbors
      
  
      
      memset(FL,0,sizeof(REAL)*NVAR_R);
      memset(FR,0,sizeof(REAL)*NVAR_R);

      // Getting the original state ===========================
      
      curcell=&(stencil[i].oct[ioct[6]].cell[icell].rfield);
      
      /* // "MUSCL-LIKE" STATE RECONSTRUCTION */
      memset(ffact,0,sizeof(int)*2);
      for(iface=0;iface<2;iface++){
	memcpy(RC+iface,curcell,sizeof(struct Rtype));
      }

      // Neighbor "MUSCL-LIKE" reconstruction
      for(iface=0;iface<2;iface++){
	inei=iface+2;
	memcpy(RN+iface,&(stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].rfield),sizeof(struct Rtype));

	int condsplit;
#ifdef COARSERAD
	condsplit=1;
#else
	condsplit=(!stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].split);
#endif
	if(condsplit){
	  ffact[iface]=1; // we cancel the contriubtion of split neighbors
	}
	
      }




      // Y DIRECTION =========================================================================
      
      // --------- solving the Riemann Problems FRONT


      
      for(igrp=0;igrp<NGRP;igrp++){
	
	// E
	up=RC[0].e[igrp];
	um=RN[0].e[igrp];

	fp=RC[0].fy[igrp];
	fm=RN[0].fy[igrp];

	FL[0+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);
	
	//FX

	up=RC[0].fx[igrp];
	um=RN[0].fx[igrp];

	fp=dEddington(RC[0].fx[igrp],RC[0].fy[igrp],RC[0].fz[igrp],RC[0].e[igrp],c,0,1);
	fm=dEddington(RN[0].fx[igrp],RN[0].fy[igrp],RN[0].fz[igrp],RN[0].e[igrp],c,0,1);

	FL[1+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

	//FY

	up=RC[0].fy[igrp];
	um=RN[0].fy[igrp];

	fp=dEddington(RC[0].fx[igrp],RC[0].fy[igrp],RC[0].fz[igrp],RC[0].e[igrp],c,1,1);
	fm=dEddington(RN[0].fx[igrp],RN[0].fy[igrp],RN[0].fz[igrp],RN[0].e[igrp],c,1,1);

	//if(up==1.) printf("FL=%e fp=%e fm=%e || ",0.5*(fp+fm)+0.5*c*(um-up),fp,fm);

	FL[2+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

	//FX

	up=RC[0].fz[igrp];
	um=RN[0].fz[igrp];

	fp=dEddington(RC[0].fx[igrp],RC[0].fy[igrp],RC[0].fz[igrp],RC[0].e[igrp],c,2,1);
	fm=dEddington(RN[0].fx[igrp],RN[0].fy[igrp],RN[0].fz[igrp],RN[0].e[igrp],c,2,1);

	FL[3+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

      }
      

	// ===========================================


      

      // --------- solving the Riemann Problems BACK

      for(igrp=0;igrp<NGRP;igrp++){
	
	// E
	up=RN[1].e[igrp];
	um=RC[1].e[igrp];

	fp=RN[1].fy[igrp];
	fm=RC[1].fy[igrp];

	FR[0+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);
	
	//FX

	up=RN[1].fx[igrp];
	um=RC[1].fx[igrp];

	fp=dEddington(RN[1].fx[igrp],RN[1].fy[igrp],RN[1].fz[igrp],RN[1].e[igrp],c,0,1);
	fm=dEddington(RC[1].fx[igrp],RC[1].fy[igrp],RC[1].fz[igrp],RC[1].e[igrp],c,0,1);

	FR[1+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

	//FY

	up=RN[1].fy[igrp];
	um=RC[1].fy[igrp];

	fp=dEddington(RN[1].fx[igrp],RN[1].fy[igrp],RN[1].fz[igrp],RN[1].e[igrp],c,1,1);
	fm=dEddington(RC[1].fx[igrp],RC[1].fy[igrp],RC[1].fz[igrp],RC[1].e[igrp],c,1,1);

	//printf("FR=%e fp=%e fm=%e up=%e um=%e\n",0.5*(fp+fm+c*(um-up)),fp,fm, up,um);
	FR[2+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

	//FX

	up=RN[1].fz[igrp];
	um=RC[1].fz[igrp];

	fp=dEddington(RN[1].fx[igrp],RN[1].fy[igrp],RN[1].fz[igrp],RN[1].e[igrp],c,2,1);
	fm=dEddington(RC[1].fx[igrp],RC[1].fy[igrp],RC[1].fz[igrp],RC[1].e[igrp],c,2,1);

	FR[3+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

      }
      
      
      //========================= copy the fluxes
      // Cancelling the fluxes from splitted neighbours

      for(igrp=0;igrp<NGRP;igrp++){
	for(iface=0;iface<NVAR_R;iface++) FL[iface+igrp*NVAR_R]*=ffact[0]; 
	for(iface=0;iface<NVAR_R;iface++) FR[iface+igrp*NVAR_R]*=ffact[1]; 
      }
      
      memcpy(stencil[i].New.cell[icell].rflux+2*NVAR_R*NGRP,FL,sizeof(REAL)*NVAR_R*NGRP);
      memcpy(stencil[i].New.cell[icell].rflux+3*NVAR_R*NGRP,FR,sizeof(REAL)*NVAR_R*NGRP);

    //ready for the next oct
  }
}
}


// ===================================================================================================

__global__ void drad_sweepZ(struct RGRID *stencil, int level, int curcpu, int nread,int stride,REAL dx, REAL dt, REAL c){

  int inei,icell,iface;
  int i,igrp;
  int vnei[6],vcell[6];

  REAL FL[NVAR_R*NGRP],FR[NVAR_R*NGRP];

  struct Rtype RC[2];
  struct Rtype RN[2];

  int ioct[7]={0,1,2,3,4,5,6};

  struct Rtype *curcell;

  int ffact[2]={0,0};
  REAL fp,fm;
  REAL up,um;
  
  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
  
  i=bx*blockDim.x+tx;
  if(i<nread){
    for(icell=0;icell<8;icell++){ // we scan the cells
      getcellnei_gpu_rad(icell, vnei, vcell); // we get the neighbors
      
      
      memset(FL,0,sizeof(REAL)*NVAR_R);
      memset(FR,0,sizeof(REAL)*NVAR_R);

      // Getting the original state ===========================
      
      curcell=&(stencil[i].oct[ioct[6]].cell[icell].rfield);
      
      /* // "MUSCL-LIKE" STATE RECONSTRUCTION */
      memset(ffact,0,sizeof(int)*2);
      for(iface=0;iface<2;iface++){
	memcpy(RC+iface,curcell,sizeof(struct Rtype));
      }

      // Neighbor "MUSCL-LIKE" reconstruction
      for(iface=0;iface<2;iface++){
	inei=iface+4;
	memcpy(RN+iface,&(stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].rfield),sizeof(struct Rtype));

	int condsplit;
#ifdef COARSERAD
	condsplit=1;
#else
	condsplit=(!stencil[i].oct[ioct[vnei[inei]]].cell[vcell[inei]].split);
#endif
	if(condsplit){
	  ffact[iface]=1; // we consider the contriubtion of split neighbors
	}
      }



      
      // Z DIRECTION =========================================================================
      
      // --------- solving the Riemann Problems BOTTOM


      
      for(igrp=0;igrp<NGRP;igrp++){
	
	// E
	up=RC[0].e[igrp];
	um=RN[0].e[igrp];

	fp=RC[0].fz[igrp];
	fm=RN[0].fz[igrp];

	FL[0+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);
	//FX

	up=RC[0].fx[igrp];
	um=RN[0].fx[igrp];

	fp=dEddington(RC[0].fx[igrp],RC[0].fy[igrp],RC[0].fz[igrp],RC[0].e[igrp],c,0,2);
	fm=dEddington(RN[0].fx[igrp],RN[0].fy[igrp],RN[0].fz[igrp],RN[0].e[igrp],c,0,2);

	FL[1+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

	//FY

	up=RC[0].fy[igrp];
	um=RN[0].fy[igrp];

	fp=dEddington(RC[0].fx[igrp],RC[0].fy[igrp],RC[0].fz[igrp],RC[0].e[igrp],c,1,2);
	fm=dEddington(RN[0].fx[igrp],RN[0].fy[igrp],RN[0].fz[igrp],RN[0].e[igrp],c,1,2);

	FL[2+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

	//FZ

	up=RC[0].fz[igrp];
	um=RN[0].fz[igrp];

	fp=dEddington(RC[0].fx[igrp],RC[0].fy[igrp],RC[0].fz[igrp],RC[0].e[igrp],c,2,2);
	fm=dEddington(RN[0].fx[igrp],RN[0].fy[igrp],RN[0].fz[igrp],RN[0].e[igrp],c,2,2);

	FL[3+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

      }
      

      // ===========================================


      

      // --------- solving the Riemann Problems TOP

      for(igrp=0;igrp<NGRP;igrp++){
	
	// E
	up=RN[1].e[igrp];
	um=RC[1].e[igrp];

	fp=RN[1].fz[igrp];
	fm=RC[1].fz[igrp];

	FR[0+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);
	
	//FX

	up=RN[1].fx[igrp];
	um=RC[1].fx[igrp];

	fp=dEddington(RN[1].fx[igrp],RN[1].fy[igrp],RN[1].fz[igrp],RN[1].e[igrp],c,0,2);
	fm=dEddington(RC[1].fx[igrp],RC[1].fy[igrp],RC[1].fz[igrp],RC[1].e[igrp],c,0,2);

	FR[1+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

	//FY

	up=RN[1].fy[igrp];
	um=RC[1].fy[igrp];

	fp=dEddington(RN[1].fx[igrp],RN[1].fy[igrp],RN[1].fz[igrp],RN[1].e[igrp],c,1,2);
	fm=dEddington(RC[1].fx[igrp],RC[1].fy[igrp],RC[1].fz[igrp],RC[1].e[igrp],c,1,2);

	FR[2+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

	//FX

	up=RN[1].fz[igrp];
	um=RC[1].fz[igrp];

	fp=dEddington(RN[1].fx[igrp],RN[1].fy[igrp],RN[1].fz[igrp],RN[1].e[igrp],c,2,2);
	fm=dEddington(RC[1].fx[igrp],RC[1].fy[igrp],RC[1].fz[igrp],RC[1].e[igrp],c,2,2);

	FR[3+igrp*NVAR_R]=0.5*(fp+fm)+0.5*c*(um-up);

      }
      
      
      //========================= copy the fluxes
      // Cancelling the fluxes from splitted neighbours

      for(igrp=0;igrp<NGRP;igrp++){
	for(iface=0;iface<NVAR_R;iface++) FL[iface+igrp*NVAR_R]*=ffact[0]; 
	for(iface=0;iface<NVAR_R;iface++) FR[iface+igrp*NVAR_R]*=ffact[1]; 
      }



      memcpy(stencil[i].New.cell[icell].rflux+4*NVAR_R*NGRP,FL,sizeof(REAL)*NVAR_R*NGRP);
      memcpy(stencil[i].New.cell[icell].rflux+5*NVAR_R*NGRP,FR,sizeof(REAL)*NVAR_R*NGRP);

      //ready for the next oct
    }
}
}

// ===================================================================================================
// ===================================================================================================

__global__ void dupdatefieldrad(struct RGRID *stencil, int nread, int stride, struct CPUINFO *cpu, REAL dxcur, REAL dtnew, REAL cloc)
{
  int i,icell,igrp;
  struct Rtype R;
  struct Rtype Rupdate;
  REAL one;
  int flx;
  REAL dtsurdx=dtnew/dxcur;
  REAL F[NFLUX_R];

  unsigned int bx=blockIdx.x;
  unsigned int tx=threadIdx.x;
  i=bx*blockDim.x+tx;
  if(i<nread){
    for(icell=0;icell<8;icell++){ // we scan the cells
      int condsplit;
#ifdef COARSERAD
      condsplit=0;
#else
      condsplit=(stencil[i].oct[6].cell[icell].split);
#endif
      if(condsplit) continue;
      memcpy(F,stencil[i].New.cell[icell].rflux,sizeof(REAL)*NFLUX_R);// New fluxes from the stencil
    
    // ==== updating
    
    // actually we compute and store the delta U only
      one=1.;
      memset(&R,0,sizeof(struct Rtype)); // setting delta U
      for(flx=0;flx<6;flx++){
	for(igrp=0;igrp<NGRP;igrp++){
	  R.e[igrp]  +=F[0+igrp*NVAR_R+flx*NVAR_R*NGRP]*dtsurdx*one;
	  R.fx[igrp] +=F[1+igrp*NVAR_R+flx*NVAR_R*NGRP]*dtsurdx*one;
	  R.fy[igrp] +=F[2+igrp*NVAR_R+flx*NVAR_R*NGRP]*dtsurdx*one;
	  R.fz[igrp] +=F[3+igrp*NVAR_R+flx*NVAR_R*NGRP]*dtsurdx*one;
	}
	one*=-1.;
      }
    
#ifndef WCHEM
      // adding the source contribution
      REAL SRC;
      for(igrp=0;igrp<NGRP;igrp++){
	SRC=stencil[i].oct[6].cell[icell].rfield.src;
	R.e[igrp]  +=SRC*dtnew+EMIN;
      }
#endif
    
    // scatter back the delta Uwithin the stencil
    
    //memcpy(&(stencil[i].New.cell[icell].deltaR),&R,sizeof(struct Rtype));
    
    // TESTING FULL UPDATE IN STENCIL APPROACH
    
    memcpy(&Rupdate,&stencil[i].New.cell[icell].rfieldnew,sizeof(struct Rtype));
    
    for(igrp=0;igrp<NGRP;igrp++){ 
      Rupdate.e[igrp]   +=R.e[igrp];
      Rupdate.fx[igrp]  +=R.fx[igrp];
      Rupdate.fy[igrp]  +=R.fy[igrp];
      Rupdate.fz[igrp]  +=R.fz[igrp];
    }
      
    //memcpy(&(curoct->cell[icell].rfieldnew),&Rupdate,sizeof(struct Rtype));
    memcpy(&stencil[i].New.cell[icell].rfieldnew,&Rupdate,sizeof(struct Rtype));
    }
  }
}

// ====================================================================================================================

int advanceradGPU (struct OCT **firstoct, int level, struct CPUINFO *cpu, struct RGRID *stencil, int stride, REAL dxcur, REAL dtnew,REAL aexp, struct RUNPARAMS *param, int chemonly){

  struct OCT *nextoct;
  struct OCT *curoct;
  struct OCT *curoct0;
  int nreadtot,nread;
  //double t[10];
  //double tg=0.,th=0.,tu=0.,ts=0.;//,tfu=0.,ttot=0.;
  REAL cloc; // the speed of light in code units
  CUDA_CHECK_ERROR("Rad Start");

  cloc=aexp*param->clight*LIGHT_SPEED_IN_M_PER_S/param->unit.unit_v;
  //printf("cloc=%e aexp=%e\n",cloc,aexp);


  cudaStream_t stream[cpu->nstream];
  int vnread[cpu->nstream];
  int is,offset;
  // creating the streams
  for(is=0;is<cpu->nstream;is++){
    cudaStreamCreate(&stream[is]);
  }

  // --------------- setting the first oct of the level
  nextoct=firstoct[level-1];
  nreadtot=0;
  int ng;
  int nt;

  if((nextoct!=NULL)&&(cpu->noct[level-1]!=0)){
    do{
      curoct0=nextoct;
      curoct=curoct0;

      //t[0]=MPI_Wtime();

      // streaming ====================
      offset=0;
      for(is=0;is<cpu->nstream;is++){
	// ------------ gathering the stencil value values
	curoct=nextoct;
	if(curoct!=NULL){
	  //printf("Start Error  -2fg=%s is=%d vnread=%d offset=%d\n",cudaGetErrorString(cudaGetLastError()),is,vnread[is],offset);
	  nextoct= gatherstencilrad(curoct,stencil+offset,stride/cpu->nstream,cpu, vnread+is,cloc);
	  //printf("Start Error  -1=%s is=%d vnread=%d offset=%d\n",cudaGetErrorString(cudaGetLastError()),is,vnread[is],offset);
	  if(vnread[is]!=0){

	    ng=((vnread[is]-1)/cpu->nthread)+1; // +1 to treat leftovers
	    if(ng==1){
	      nt=vnread[is];
	    }
	    else{
	      nt=cpu->nthread;
	    }
	  
	    dim3 gridoct(ng);
	    dim3 blockoct(nt);
	  
#ifdef WCHEM
	    dim3 gridoct_chem(ng);
	    dim3 blockoct_chem(nt);
#endif      

	    //t[2]=MPI_Wtime();
	  
	    //printf("Start Error  0=%s is=%d vnread=%d offset=%d\n",cudaGetErrorString(cudaGetLastError()),is,vnread[is],offset);

	    cudaMemcpyAsync(cpu->rad_stencil+offset,stencil+offset,vnread[is]*sizeof(struct RGRID),cudaMemcpyHostToDevice,stream[is]);  
	  
	    //printf("Start Error  1=%s is=%d vnread=%d offset=%d\n",cudaGetErrorString(cudaGetLastError()),is,vnread[is],offset);
	  
#ifndef COARSERAD
	    int condadvec=1;
#else
	    int condadvec=((level==param->lcoarse)&&(!chemonly));
#endif
	    

#ifndef NOCOMP
	    if(condadvec){
/* // ------------ solving the hydro */
	      drad_sweepX<<<gridoct,blockoct,0,stream[is]>>>(cpu->rad_stencil+offset,level,cpu->rank,vnread[is],stride,dxcur,dtnew,cloc);   
	      drad_sweepY<<<gridoct,blockoct,0,stream[is]>>>(cpu->rad_stencil+offset,level,cpu->rank,vnread[is],stride,dxcur,dtnew,cloc);  
	      drad_sweepZ<<<gridoct,blockoct,0,stream[is]>>>(cpu->rad_stencil+offset,level,cpu->rank,vnread[is],stride,dxcur,dtnew,cloc);  
	    }
	  
	    //printf("Start Error  2=%s\n",cudaGetErrorString(cudaGetLastError()));
	    // ------------ updating values within the stencil
	  
	    //t[4]=MPI_Wtime();
	  
	    if(condadvec) dupdatefieldrad<<<gridoct,blockoct,0,stream[is]>>>(cpu->rad_stencil+offset,vnread[is],stride,cpu,dxcur,dtnew,cloc); 
	    //printf("Start Error  3=%s\n",cudaGetErrorString(cudaGetLastError()));
	    // ----------- perform physical cooling and ionisation 
#ifdef WCHEM
	    dchemrad<<<gridoct_chem,blockoct_chem,0,stream[is]>>>(cpu->rad_stencil+offset,vnread[is],stride,cpu,dxcur,dtnew,cpu->dparam,aexp,chemonly); 
#endif
	    
#endif
	    cudaMemcpyAsync(stencil+offset,cpu->rad_stencil+offset,vnread[is]*sizeof(struct RGRID),cudaMemcpyDeviceToHost,stream[is]);
	    //printf("Start Error  4=%s\n",cudaGetErrorString(cudaGetLastError()));

	    offset+=vnread[is];
	  }
	}
      }
      
      // ------------ scatter back the FLUXES
      //CUDA_CHECK_ERROR("Bef Sync");
      cudaDeviceSynchronize();
      //CUDA_CHECK_ERROR("Af Sync");
      //t[6]=MPI_Wtime();
   
      nread=offset;
      //printf("Start Error  5=%s\n",cudaGetErrorString(cudaGetLastError()));
      nextoct=scatterstencilrad(curoct0,stencil, nread, cpu,dxcur,dtnew,cloc);

      //printf("Start Error  6=%s\n",cudaGetErrorString(cudaGetLastError()));

      //t[8]=MPI_Wtime();

      nreadtot+=nread;
    }while(nextoct!=NULL);
  }

  // Destroying the streams
  for(is=0;is<cpu->nstream;is++){
    cudaStreamDestroy(stream[is]);
  }
  //printf("Start Error Hyd =%s nreadtot=%d\n",cudaGetErrorString(cudaGetLastError()),nreadtot);

  //printf("GPU | tgat=%e tcal=%e tup=%e tscat=%e\n",tg,th,tu,ts);
  CUDA_CHECK_ERROR("Rad Stop");

  return nreadtot;
}

#endif
